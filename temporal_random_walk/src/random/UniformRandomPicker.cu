#include "UniformRandomPicker.cuh"

#include <stdexcept>
#include "../utils/rand_utils.cuh"

template<GPUUsageMode GPUUsage>
int UniformRandomPicker<GPUUsage>::pick_random_host(const int start, const int end, const bool prioritize_end) {
    if (start >= end) {
        throw std::invalid_argument("Start must be less than end.");
    }

    return generate_random_int_host(start, end - 1);
}

#ifdef HAS_CUDA
template<GPUUsageMode GPUUsage>
int UniformRandomPicker<GPUUsage>::pick_random_device(const int start, const int end, const bool prioritize_end, hiprandState* rand_state) {
    if (start >= end) {
        return -1;
    }

    return generate_random_int_device(start, end - 1, rand_state);
}
#endif

template class UniformRandomPicker<GPUUsageMode::ON_CPU>;
#ifdef HAS_CUDA
template class UniformRandomPicker<GPUUsageMode::ON_GPU>;
#endif
