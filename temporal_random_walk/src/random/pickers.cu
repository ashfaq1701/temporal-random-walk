#include "hip/hip_runtime.h"
#include "pickers.cuh"

#include <cmath>
#include <stdexcept>

#ifdef HAS_CUDA
#include <cuda/std/__algorithm/lower_bound.h>
#endif

#include "../utils/random.cuh"

HOST DEVICE int random_pickers::pick_random_linear(const int start, const int end, const bool prioritize_end, const double rand_number) {
    if (start >= end) {
        return -1;
    }

    const int len_seq = end - start;

    // For a sequence of length n, weights form an arithmetic sequence
    // When prioritizing end: weights are 1, 2, 3, ..., n
    // When prioritizing start: weights are n, n-1, n-2, ..., 1
    // Sum of arithmetic sequence = n(a1 + an)/2 = n(n+1)/2
    const double total_weight = static_cast<double>(len_seq) *
                                   (static_cast<double>(len_seq) + 1.0) / 2.0;

    // Generate random value in [0, total_weight)
    const double scaled_random_value = total_weight * rand_number;

    // For both cases, we solve quadratic equation i² + i - 2r = 0
    // where r is our random value (or transformed random value)
    // Using quadratic formula: (-1 ± √(1 + 8r))/2
    const double discriminant = 1.0 + 8.0 * scaled_random_value;
    const double root = (-1.0 + std::sqrt(discriminant)) / 2.0;
    const int index = static_cast<int>(std::floor(root));

    if (prioritize_end) {
        // For prioritize_end=true, larger indices should have higher probability
        return start + std::min(index, len_seq - 1);
    } else {
        // For prioritize_end=false, we reverse the index to give
        // higher probability to smaller indices
        const int revered_index = len_seq - 1 - index;
        return start + std::max(0, revered_index);
    }
}

HOST DEVICE int random_pickers::pick_random_exponential_index(const int start, const int end, const bool prioritize_end, const double rand_number) {
    if (start >= end) {
        return -1;
    }

    const int len_seq = end - start;

    double k;
    if (len_seq < 710) {
        // Inverse CDF formula,
        // k = ln(1 + u * (e^len seq − 1)) − 1
        k = log1p(rand_number * expm1(len_seq)) - 1;
    } else {
        // Inverse CDF approximation for large len_seq,
        // k = len_seq + ln(u) − 1
        k = len_seq + std::log(rand_number) - 1;
    }

    // Due to rounding, the trailing "-1" in the inverse CDF formula causes error.
    // To compensate for this we add 1 with k.
    // And bound the results within limits.
    const int rounded_index = std::max(0, std::min(static_cast<int>(k + 1), len_seq - 1));

    if (prioritize_end) {
        return start + rounded_index;
    } else {
        return start + (len_seq - 1 - rounded_index);
    }
}

HOST DEVICE int random_pickers::pick_random_uniform(const int start, const int end, const double rand_number) {
    if (start >= end) {
        return -1;
    }

    return start + static_cast<int>(rand_number * (end - start));
}

HOST int random_pickers::pick_random_exponential_weights_host(double* weights, const size_t weights_size, const size_t group_start, const size_t group_end, const double random_number) {
    if (group_start >= group_end || group_end > weights_size) {
        return -1;
    }

    // Get start and end sums
    double start_sum = 0.0;
    if (group_start > 0) {
        start_sum = weights[group_start - 1];
    }
    const double end_sum = weights[group_end - 1];

    if (end_sum < start_sum) {
        return -1;
    }

    const double random_val = start_sum + random_number * (end_sum - start_sum);

    return static_cast<int>(std::lower_bound(
            weights + group_start,
            weights + group_end,
            random_val) - weights);
}

#ifdef HAS_CUDA

DEVICE int random_pickers::pick_random_exponential_weights_device(double* weights, const size_t weights_size, const size_t group_start, const size_t group_end, const double random_number) {
    if (group_start >= group_end || group_end > weights_size) {
        return -1;
    }

    // Get start and end sums
    double start_sum = 0.0;
    if (group_start > 0) {
        start_sum = weights[group_start - 1];
    }
    const double end_sum = weights[group_end - 1];

    if (end_sum < start_sum) {
        return -1;
    }

    const double random_val = start_sum + random_number * (end_sum - start_sum);

    return static_cast<int>(cuda::std::lower_bound(
            weights + group_start,
            weights + group_end,
            random_val) - weights);
}


#endif

HOST DEVICE bool random_pickers::is_index_based_picker(const RandomPickerType picker_type) {
    return picker_type == RandomPickerType::Linear || picker_type == RandomPickerType::Uniform ||
        picker_type == RandomPickerType::ExponentialIndex ||
            // ONLY FOR TESTS
            picker_type == RandomPickerType::TEST_FIRST || picker_type == RandomPickerType::TEST_LAST;
}

HOST DEVICE int random_pickers::pick_using_index_based_picker(const RandomPickerType random_picker, const int start, const int end, const bool prioritize_end, const double random_number) {
    switch (random_picker) {
        case RandomPickerType::Linear:
            return pick_random_linear(start, end, prioritize_end, random_number);
        case RandomPickerType::ExponentialIndex:
            return pick_random_exponential_index(start, end, prioritize_end, random_number);
        case RandomPickerType::Uniform:
            return pick_random_uniform(start, end, random_number);
        // ONLY FOR TEST
        case RandomPickerType::TEST_FIRST:
            return start;
        case RandomPickerType::TEST_LAST:
            return end - 1;
        default:
            return -1;
    }
}

HOST int random_pickers::pick_using_weight_based_picker_host(const RandomPickerType random_picker, double* weights, const size_t weights_size, const size_t group_start, const size_t group_end, const double random_number) {
    if (random_picker != RandomPickerType::ExponentialWeight) {
        return -1;
    }

    return pick_random_exponential_weights_host(weights, weights_size, group_start, group_end, random_number);
}

#ifdef HAS_CUDA

DEVICE int random_pickers::pick_using_weight_based_picker_device(const RandomPickerType random_picker, double* weights, const size_t weights_size, const size_t group_start, const size_t group_end, const double random_number) {
    if (random_picker != RandomPickerType::ExponentialWeight) {
        return -1;
    }

    return pick_random_exponential_weights_device(weights, weights_size, group_start, group_end, random_number);
}

#endif
