#include "hip/hip_runtime.h"
#include "node_edge_index.cuh"

#ifdef HAS_CUDA
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#endif

#include <omp.h>
#include <cmath>
#include <algorithm>
#include "../utils/omp_utils.cuh"
#include "../common/parallel_algorithms.cuh"
#include "../common/cuda_config.cuh"

/**
 * Common Functions
 */

HOST void node_edge_index::clear(NodeEdgeIndexStore *node_edge_index) {
    // Clear edge CSR structures
    clear_memory(&node_edge_index->outbound_offsets, node_edge_index->use_gpu);
    node_edge_index->outbound_offsets_size = 0;

    clear_memory(&node_edge_index->outbound_indices, node_edge_index->use_gpu);
    node_edge_index->outbound_indices_size = 0;

    clear_memory(&node_edge_index->outbound_timestamp_group_offsets, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_offsets_size = 0;

    clear_memory(&node_edge_index->outbound_timestamp_group_indices, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_indices_size = 0;

    // Clear inbound structures
    clear_memory(&node_edge_index->inbound_offsets, node_edge_index->use_gpu);
    node_edge_index->inbound_offsets_size = 0;

    clear_memory(&node_edge_index->inbound_indices, node_edge_index->use_gpu);
    node_edge_index->inbound_indices_size = 0;

    clear_memory(&node_edge_index->inbound_timestamp_group_offsets, node_edge_index->use_gpu);
    node_edge_index->inbound_timestamp_group_offsets_size = 0;

    clear_memory(&node_edge_index->inbound_timestamp_group_indices, node_edge_index->use_gpu);
    node_edge_index->inbound_timestamp_group_indices_size = 0;

    // Clear temporal weights
    clear_memory(&node_edge_index->outbound_forward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = 0;

    clear_memory(&node_edge_index->outbound_backward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = 0;

    clear_memory(&node_edge_index->inbound_backward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->inbound_backward_cumulative_weights_exponential_size = 0;
}

HOST DEVICE SizeRange node_edge_index::get_edge_range(const NodeEdgeIndexStore *node_edge_index, const int dense_node_id,
                                            const bool forward, const bool is_directed) {
    if (is_directed) {
        const size_t *offsets = forward ? node_edge_index->outbound_offsets : node_edge_index->inbound_offsets;
        size_t offsets_size = forward ? node_edge_index->outbound_offsets_size : node_edge_index->inbound_offsets_size;

        if (dense_node_id < 0 || dense_node_id >= offsets_size - 1) {
            return SizeRange{0, 0};
        }

        const size_t start = offsets[dense_node_id];
        const size_t end = offsets[dense_node_id + 1];

        return SizeRange{start, end};
    } else {
        if (dense_node_id < 0 || dense_node_id >= node_edge_index->outbound_offsets_size - 1) {
            return SizeRange{0, 0};
        }

        const size_t start = node_edge_index->outbound_offsets[dense_node_id];
        const size_t end = node_edge_index->outbound_offsets[dense_node_id + 1];

        return SizeRange{start, end};
    }
}

HOST DEVICE SizeRange node_edge_index::get_timestamp_group_range(const NodeEdgeIndexStore *node_edge_index,
                                                       const int dense_node_id, const size_t group_idx,
                                                       const bool forward, const bool is_directed) {
    const size_t *group_offsets = nullptr;
    size_t group_offsets_size = 0;
    const size_t *group_indices = nullptr;
    const size_t *edge_offsets = nullptr;

    if (is_directed && !forward) {
        group_offsets = node_edge_index->inbound_timestamp_group_offsets;
        group_offsets_size = node_edge_index->inbound_timestamp_group_offsets_size;
        group_indices = node_edge_index->inbound_timestamp_group_indices;
        edge_offsets = node_edge_index->inbound_offsets;
    } else {
        group_offsets = node_edge_index->outbound_timestamp_group_offsets;
        group_offsets_size = node_edge_index->outbound_timestamp_group_offsets_size;
        group_indices = node_edge_index->outbound_timestamp_group_indices;
        edge_offsets = node_edge_index->outbound_offsets;
    }

    if (dense_node_id < 0 || dense_node_id >= group_offsets_size - 1) {
        return SizeRange{0, 0};
    }

    const size_t node_group_start = group_offsets[dense_node_id];
    const size_t node_group_end = group_offsets[dense_node_id + 1];

    const size_t num_groups = node_group_end - node_group_start;
    if (group_idx >= num_groups) {
        return SizeRange{0, 0};
    }

    const size_t group_start_idx = node_group_start + group_idx;
    const size_t group_start = group_indices[group_start_idx];

    // Group end is either next group's start or node's edge range end
    size_t group_end = 0;
    if (group_idx == num_groups - 1) {
        group_end = edge_offsets[dense_node_id + 1];
    } else {
        group_end = group_indices[group_start_idx + 1];
    }

    return SizeRange{group_start, group_end};
}

HOST DEVICE MemoryView<size_t> node_edge_index::get_timestamp_offset_vector(const NodeEdgeIndexStore *node_edge_index,
                                                                  const bool forward, const bool is_directed) {
    if (is_directed && !forward) {
        return MemoryView<size_t>{
            node_edge_index->inbound_timestamp_group_offsets,
            node_edge_index->inbound_timestamp_group_offsets_size
        };
    } else {
        return MemoryView<size_t>{
            node_edge_index->outbound_timestamp_group_offsets,
            node_edge_index->outbound_timestamp_group_offsets_size
        };
    }
}

HOST DEVICE size_t node_edge_index::get_timestamp_group_count(const NodeEdgeIndexStore *node_edge_index, const int dense_node_id,
                                                    const bool forward, const bool is_directed) {
    // Get the appropriate timestamp offset vector
    MemoryView<size_t> offsets_block = get_timestamp_offset_vector(node_edge_index, forward, is_directed);
    const size_t *offsets = offsets_block.data;
    size_t offsets_size = offsets_block.size;

    // Check if the node ID is valid
    if (dense_node_id < 0 || dense_node_id >= offsets_size - 1) {
        return 0;
    }

    // Get start and end offsets for the node
    const size_t start = offsets[dense_node_id];
    const size_t end = offsets[dense_node_id + 1];

    return end - start;
}

/**
 * Rebuild related functions
 */

HOST void node_edge_index::allocate_node_edge_offsets(NodeEdgeIndexStore *node_edge_index, const size_t node_index_capacity,
                                            const bool is_directed) {
    allocate_memory(&node_edge_index->outbound_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
    node_edge_index->outbound_offsets_size = node_index_capacity + 1;
    fill_memory(node_edge_index->outbound_offsets, node_index_capacity + 1, static_cast<size_t>(0),
                node_edge_index->use_gpu);

    allocate_memory(&node_edge_index->outbound_timestamp_group_offsets, node_index_capacity + 1,
                    node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_offsets_size = node_index_capacity + 1;
    fill_memory(node_edge_index->outbound_timestamp_group_offsets, node_index_capacity + 1, static_cast<size_t>(0),
                node_edge_index->use_gpu);

    // For directed graphs, also allocate inbound structures
    if (is_directed) {
        allocate_memory(&node_edge_index->inbound_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
        node_edge_index->inbound_offsets_size = node_index_capacity + 1;
        fill_memory(node_edge_index->inbound_offsets, node_index_capacity + 1, static_cast<size_t>(0),
                    node_edge_index->use_gpu);

        allocate_memory(&node_edge_index->inbound_timestamp_group_offsets, node_index_capacity + 1,
                        node_edge_index->use_gpu);
        node_edge_index->inbound_timestamp_group_offsets_size = node_index_capacity + 1;
        fill_memory(node_edge_index->inbound_timestamp_group_offsets, node_index_capacity + 1, static_cast<size_t>(0),
                    node_edge_index->use_gpu);
    }
}

HOST void node_edge_index::allocate_node_edge_indices(NodeEdgeIndexStore *node_edge_index, const bool is_directed) {
    size_t num_outbound_edges = 0;

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        // For GPU memory, we need to copy the value back to host
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_outbound_edges,
            node_edge_index->outbound_offsets + (node_edge_index->outbound_offsets_size - 1),
            sizeof(size_t),
            hipMemcpyDeviceToHost));
    } else
    #endif
    {
        // For CPU memory, we can access it directly
        num_outbound_edges = node_edge_index->outbound_offsets[node_edge_index->outbound_offsets_size - 1];
    }

    // Allocate memory for outbound indices
    allocate_memory(&node_edge_index->outbound_indices, num_outbound_edges, node_edge_index->use_gpu);
    node_edge_index->outbound_indices_size = num_outbound_edges;

    // For directed graphs, also allocate inbound indices
    if (is_directed) {
        size_t num_inbound_edges = 0;

        #ifdef HAS_CUDA
        if (node_edge_index->use_gpu) {
            CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_inbound_edges,
                node_edge_index->inbound_offsets + (node_edge_index->inbound_offsets_size - 1),
                sizeof(size_t),
                hipMemcpyDeviceToHost));
        } else
        #endif
        {
            num_inbound_edges = node_edge_index->inbound_offsets[node_edge_index->inbound_offsets_size - 1];
        }

        allocate_memory(&node_edge_index->inbound_indices, num_inbound_edges, node_edge_index->use_gpu);
        node_edge_index->inbound_indices_size = num_inbound_edges;
    }
}

HOST void node_edge_index::allocate_node_timestamp_indices(NodeEdgeIndexStore *node_edge_index, const bool is_directed) {
    size_t num_outbound_groups = 0;

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        // For GPU memory, we need to copy the value back to host
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_outbound_groups,
            node_edge_index->outbound_timestamp_group_offsets + (node_edge_index->outbound_timestamp_group_offsets_size
                - 1),
            sizeof(size_t),
            hipMemcpyDeviceToHost));
    } else
    #endif
    {
        // For CPU memory, we can access it directly
        num_outbound_groups = node_edge_index->outbound_timestamp_group_offsets[
            node_edge_index->outbound_timestamp_group_offsets_size - 1];
    }

    // Allocate memory for outbound timestamp group indices
    allocate_memory(&node_edge_index->outbound_timestamp_group_indices, num_outbound_groups, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_indices_size = num_outbound_groups;

    // For directed graphs, also allocate inbound timestamp group indices
    if (is_directed) {
        size_t num_inbound_groups = 0;

        #ifdef HAS_CUDA
        if (node_edge_index->use_gpu) {
            CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_inbound_groups,
                node_edge_index->inbound_timestamp_group_offsets + (node_edge_index->
                    inbound_timestamp_group_offsets_size - 1),
                sizeof(size_t),
                hipMemcpyDeviceToHost));
        } else
        #endif
        {
            num_inbound_groups = node_edge_index->inbound_timestamp_group_offsets[
                node_edge_index->inbound_timestamp_group_offsets_size - 1];
        }

        allocate_memory(&node_edge_index->inbound_timestamp_group_indices, num_inbound_groups,
                        node_edge_index->use_gpu);
        node_edge_index->inbound_timestamp_group_indices_size = num_inbound_groups;
    }
}

/**
 * Std implementations
 */
HOST void node_edge_index::compute_node_edge_offsets_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const bool is_directed
) {
    const size_t num_edges = edge_data->timestamps_size;

    auto* outbound_offsets = node_edge_index->outbound_offsets;
    auto* inbound_offsets  = node_edge_index->inbound_offsets;
    const auto* sources    = edge_data->sources;
    const auto* targets    = edge_data->targets;

    const size_t offset_size = node_edge_index->outbound_offsets_size;

    // Step 1: Zero out offset arrays
    std::fill_n(outbound_offsets, offset_size, 0);
    if (is_directed) {
        std::fill_n(inbound_offsets, node_edge_index->inbound_offsets_size, 0);
    }

    // Step 2: Count edge occurrences (use atomic to avoid collisions)
    #pragma omp parallel for
    for (size_t i = 0; i < num_edges; ++i) {
        const int src_idx = sources[i];
        const int tgt_idx = targets[i];

        #pragma omp atomic
        outbound_offsets[src_idx + 1]++;

        if (is_directed) {
            #pragma omp atomic
            inbound_offsets[tgt_idx + 1]++;
        } else {
            #pragma omp atomic
            outbound_offsets[tgt_idx + 1]++;
        }
    }

    // Step 3: Inclusive scan over offsets[1..]
    parallel_inclusive_scan(outbound_offsets + 1, offset_size - 1);

    if (is_directed) {
        parallel_inclusive_scan(inbound_offsets + 1, node_edge_index->inbound_offsets_size - 1);
    }
}

HOST void node_edge_index::compute_node_edge_indices_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    EdgeWithEndpointType* outbound_edge_indices_buffer,
    const bool is_directed
) {
    const size_t edges_size = edge_data->timestamps_size;
    const size_t buffer_size = is_directed ? edges_size : edges_size * 2;

    const auto* sources = edge_data->sources;
    const auto* targets = edge_data->targets;

    // Step 1: Fill outbound_edge_indices_buffer
    #pragma omp parallel for
    for (size_t i = 0; i < edges_size; ++i) {
        size_t outbound_index = is_directed ? i : i * 2;
        outbound_edge_indices_buffer[outbound_index] = EdgeWithEndpointType{static_cast<long>(i), true};

        if (!is_directed) {
            outbound_edge_indices_buffer[outbound_index + 1] = EdgeWithEndpointType{static_cast<long>(i), false};
        }
    }

    // Step 2: Fill inbound_indices for directed graphs
    if (is_directed) {
        #pragma omp parallel for
        for (size_t i = 0; i < edges_size; ++i) {
            node_edge_index->inbound_indices[i] = i;
        }
    }

    // Step 3: Stable sort outbound_edge_indices_buffer by node ID (source or target depending on flag)
    parallel::stable_sort(
        outbound_edge_indices_buffer,
        outbound_edge_indices_buffer + buffer_size,
        [sources, targets](const EdgeWithEndpointType& a, const EdgeWithEndpointType& b) {
            const int node_a = a.is_source ? sources[a.edge_id] : targets[a.edge_id];
            const int node_b = b.is_source ? sources[b.edge_id] : targets[b.edge_id];
            return node_a < node_b;
        }
    );

    // Step 4: Stable sort inbound_indices by target node (if directed)
    if (is_directed) {
        parallel::stable_sort(
            node_edge_index->inbound_indices,
            node_edge_index->inbound_indices + edges_size,
            [targets](const size_t a, const size_t b) {
                return targets[a] < targets[b];
            }
        );
    }

    // Step 5: Extract edge_id from buffer to outbound_indices
    #pragma omp parallel for
    for (size_t i = 0; i < buffer_size; ++i) {
        node_edge_index->outbound_indices[i] = outbound_edge_indices_buffer[i].edge_id;
    }
}

HOST void node_edge_index::compute_node_timestamp_offsets_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_count,
    const bool is_directed
) {
    std::vector<size_t> outbound_group_count(node_count, 0);
    std::vector<size_t> inbound_group_count;

    if (is_directed) {
        inbound_group_count.resize(node_count, 0);
    }

    const int64_t* timestamps = edge_data->timestamps;
    const size_t* outbound_offsets = node_edge_index->outbound_offsets;
    const size_t* outbound_indices = node_edge_index->outbound_indices;
    const size_t* inbound_offsets = node_edge_index->inbound_offsets;
    const size_t* inbound_indices = node_edge_index->inbound_indices;

    size_t* outbound_ts_group_offsets = node_edge_index->outbound_timestamp_group_offsets;
    size_t* inbound_ts_group_offsets = node_edge_index->inbound_timestamp_group_offsets;

    // Step 1: Count timestamp groups in parallel
    #pragma omp parallel for
    for (size_t node = 0; node < node_count; ++node) {
        // Outbound groups
        size_t start = outbound_offsets[node];
        size_t end = outbound_offsets[node + 1];

        if (start < end) {
            outbound_group_count[node] = 1;
            for (size_t i = start + 1; i < end; ++i) {
                if (timestamps[outbound_indices[i]] != timestamps[outbound_indices[i - 1]]) {
                    ++outbound_group_count[node];
                }
            }
        }

        // Inbound groups (if directed)
        if (is_directed) {
            start = inbound_offsets[node];
            end = inbound_offsets[node + 1];

            if (start < end) {
                inbound_group_count[node] = 1;
                for (size_t i = start + 1; i < end; ++i) {
                    if (timestamps[inbound_indices[i]] != timestamps[inbound_indices[i - 1]]) {
                        ++inbound_group_count[node];
                    }
                }
            }
        }
    }

    // Step 2: Compute exclusive prefix sums
    parallel_exclusive_scan(
        outbound_group_count.data(),
        outbound_ts_group_offsets,
        node_count
    );

    if (is_directed) {
        parallel_exclusive_scan(
            inbound_group_count.data(),
            inbound_ts_group_offsets,
            node_count
        );
    }
}

HOST void node_edge_index::compute_node_timestamp_indices_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_index_capacity,
    const bool is_directed
) {
    const auto timestamps = edge_data->timestamps;

    const auto outbound_offsets = node_edge_index->outbound_offsets;
    const auto outbound_indices = node_edge_index->outbound_indices;
    const auto outbound_ts_group_offsets = node_edge_index->outbound_timestamp_group_offsets;
    const auto outbound_ts_group_indices = node_edge_index->outbound_timestamp_group_indices;

    const auto inbound_offsets = node_edge_index->inbound_offsets;
    const auto inbound_indices = node_edge_index->inbound_indices;
    const auto inbound_ts_group_offsets = node_edge_index->inbound_timestamp_group_offsets;
    const auto inbound_ts_group_indices = node_edge_index->inbound_timestamp_group_indices;

    // Parallel over nodes
    #pragma omp parallel for
    for (size_t node = 0; node < node_index_capacity; ++node) {
        // Outbound timestamp group indices
        size_t start = outbound_offsets[node];
        size_t end = outbound_offsets[node + 1];
        size_t group_pos = outbound_ts_group_offsets[node];

        if (start < end) {
            outbound_ts_group_indices[group_pos++] = start;
            for (size_t i = start + 1; i < end; ++i) {
                const size_t curr_id = outbound_indices[i];
                size_t prev_id = outbound_indices[i - 1];
                if (timestamps[curr_id] != timestamps[prev_id]) {
                    outbound_ts_group_indices[group_pos++] = i;
                }
            }
        }

        // Inbound timestamp group indices (if directed)
        if (is_directed) {
            start = inbound_offsets[node];
            end = inbound_offsets[node + 1];
            group_pos = inbound_ts_group_offsets[node];

            if (start < end) {
                inbound_ts_group_indices[group_pos++] = start;
                for (size_t i = start + 1; i < end; ++i) {
                    const size_t curr_id = inbound_indices[i];
                    size_t prev_id = inbound_indices[i - 1];
                    if (timestamps[curr_id] != timestamps[prev_id]) {
                        inbound_ts_group_indices[group_pos++] = i;
                    }
                }
            }
        }
    }
}

HOST void node_edge_index::update_temporal_weights_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const double timescale_bound
) {
    const size_t node_index_capacity = node_edge_index->outbound_offsets_size - 1;
    const size_t outbound_groups_size = node_edge_index->outbound_timestamp_group_indices_size;

    // Resize memory for outbound weights
    resize_memory(
        &node_edge_index->outbound_forward_cumulative_weights_exponential,
        node_edge_index->outbound_forward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = outbound_groups_size;

    resize_memory(
        &node_edge_index->outbound_backward_cumulative_weights_exponential,
        node_edge_index->outbound_backward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = outbound_groups_size;

    const bool is_directed = node_edge_index->inbound_offsets_size > 0;

    if (is_directed) {
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;
        resize_memory(
            &node_edge_index->inbound_backward_cumulative_weights_exponential,
            node_edge_index->inbound_backward_cumulative_weights_exponential_size,
            inbound_groups_size,
            node_edge_index->use_gpu
        );
        node_edge_index->inbound_backward_cumulative_weights_exponential_size = inbound_groups_size;
    }

    // Parallel over all nodes
    #pragma omp parallel for
    for (size_t node = 0; node < node_index_capacity; ++node) {
        // === Outbound ===
        auto outbound_offsets = get_timestamp_offset_vector(node_edge_index, true, false);
        const size_t out_start = outbound_offsets.data[node];
        const size_t out_end = outbound_offsets.data[node + 1];

        if (out_start < out_end) {
            const auto* ts_group_indices = node_edge_index->outbound_timestamp_group_indices;
            const auto* edge_indices = node_edge_index->outbound_indices;
            auto* f_weights = node_edge_index->outbound_forward_cumulative_weights_exponential;
            auto* b_weights = node_edge_index->outbound_backward_cumulative_weights_exponential;
            const auto* timestamps = edge_data->timestamps;

            const int64_t min_ts = timestamps[edge_indices[ts_group_indices[out_start]]];
            const int64_t max_ts = timestamps[edge_indices[ts_group_indices[out_end - 1]]];
            const auto time_diff = static_cast<double>(max_ts - min_ts);
            const double time_scale = (timescale_bound > 0 && time_diff > 0) ? timescale_bound / time_diff : 1.0;

            double forward_sum = 0.0, backward_sum = 0.0;

            for (size_t pos = out_start; pos < out_end; ++pos) {
                const size_t edge_start = ts_group_indices[pos];
                const int64_t group_ts = timestamps[edge_indices[edge_start]];

                const double f_scaled = (timescale_bound > 0) ? static_cast<double>(max_ts - group_ts) * time_scale : static_cast<double>(max_ts - group_ts);
                const double b_scaled = (timescale_bound > 0) ? static_cast<double>(group_ts - min_ts) * time_scale : static_cast<double>(group_ts - min_ts);

                const double fw = std::exp(f_scaled);
                const double bw = std::exp(b_scaled);

                f_weights[pos] = fw;
                b_weights[pos] = bw;
                forward_sum += fw;
                backward_sum += bw;
            }

            double f_cumsum = 0.0, b_cumsum = 0.0;
            for (size_t pos = out_start; pos < out_end; ++pos) {
                f_weights[pos] /= forward_sum;
                b_weights[pos] /= backward_sum;
                f_cumsum += f_weights[pos];
                b_cumsum += b_weights[pos];
                f_weights[pos] = f_cumsum;
                b_weights[pos] = b_cumsum;
            }
        }

        // === Inbound ===
        if (is_directed) {
            auto inbound_offsets = get_timestamp_offset_vector(node_edge_index, false, true);
            const size_t in_start = inbound_offsets.data[node];
            const size_t in_end = inbound_offsets.data[node + 1];

            if (in_start < in_end) {
                const auto* ts_group_indices = node_edge_index->inbound_timestamp_group_indices;
                const auto* edge_indices = node_edge_index->inbound_indices;
                auto* b_weights = node_edge_index->inbound_backward_cumulative_weights_exponential;
                const auto* timestamps = edge_data->timestamps;

                const int64_t min_ts = timestamps[edge_indices[ts_group_indices[in_start]]];
                const int64_t max_ts = timestamps[edge_indices[ts_group_indices[in_end - 1]]];
                const auto time_diff = static_cast<double>(max_ts - min_ts);
                const double time_scale = (timescale_bound > 0 && time_diff > 0) ? timescale_bound / time_diff : 1.0;

                double backward_sum = 0.0;

                for (size_t pos = in_start; pos < in_end; ++pos) {
                    const size_t edge_start = ts_group_indices[pos];
                    const int64_t group_ts = timestamps[edge_indices[edge_start]];
                    const double b_scaled = (timescale_bound > 0) ? static_cast<double>(group_ts - min_ts) * time_scale : static_cast<double>(group_ts - min_ts);
                    const double bw = std::exp(b_scaled);
                    b_weights[pos] = bw;
                    backward_sum += bw;
                }

                double b_cumsum = 0.0;
                for (size_t pos = in_start; pos < in_end; ++pos) {
                    b_weights[pos] /= backward_sum;
                    b_cumsum += b_weights[pos];
                    b_weights[pos] = b_cumsum;
                }
            }
        }
    }
}

/**
 * Cuda implementations
 */
#ifdef HAS_CUDA

HOST void node_edge_index::compute_node_edge_offsets_cuda(
    NodeEdgeIndexStore *node_edge_index,
    const EdgeDataStore *edge_data,
    bool is_directed
) {
    const size_t num_edges = edge_data->timestamps_size;

    // Get raw pointers to work with
    size_t *outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t *inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    int *src_ptr = edge_data->sources;
    int *tgt_ptr = edge_data->targets;

    // Count edges per node using atomics
    auto counter_device_lambda = [
                outbound_offsets_ptr, inbound_offsets_ptr,
                src_ptr, tgt_ptr, is_directed] DEVICE (const size_t i) {
        const int src_idx = src_ptr[i];
        const int tgt_idx = tgt_ptr[i];

        atomicAdd(reinterpret_cast<unsigned int *>(&outbound_offsets_ptr[src_idx + 1]), 1);
        if (is_directed) {
            atomicAdd(reinterpret_cast<unsigned int *>(&inbound_offsets_ptr[tgt_idx + 1]), 1);
        } else {
            atomicAdd(reinterpret_cast<unsigned int *>(&outbound_offsets_ptr[tgt_idx + 1]), 1);
        }
    };

    // Process all edges in parallel
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(num_edges),
        counter_device_lambda);
    CUDA_KERNEL_CHECK("After thrust for_each in compute_node_edge_offsets_cuda");

    // Calculate prefix sums for outbound edge offsets
    thrust::device_ptr<size_t> d_outbound_offsets(outbound_offsets_ptr);
    thrust::inclusive_scan(
        DEVICE_EXECUTION_POLICY,
        d_outbound_offsets + 1,
        d_outbound_offsets + static_cast<long>(node_edge_index->outbound_offsets_size),
        d_outbound_offsets + 1
    );
    CUDA_KERNEL_CHECK("After thrust inclusive_scan outbound in compute_node_edge_offsets_cuda");

    // Calculate prefix sums for inbound edge offsets (if directed)
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_offsets(inbound_offsets_ptr);
        thrust::inclusive_scan(
            DEVICE_EXECUTION_POLICY,
            d_inbound_offsets + 1,
            d_inbound_offsets + static_cast<long>(node_edge_index->inbound_offsets_size),
            d_inbound_offsets + 1
        );
        CUDA_KERNEL_CHECK("After thrust inclusive_scan inbound in compute_node_edge_offsets_cuda");
    }
}

HOST void node_edge_index::compute_node_edge_indices_cuda(
    NodeEdgeIndexStore *node_edge_index,
    const EdgeDataStore *edge_data,
    EdgeWithEndpointType *outbound_edge_indices_buffer,
    bool is_directed
) {
    const size_t edges_size = edge_data->timestamps_size;
    const size_t buffer_size = is_directed ? edges_size : edges_size * 2;

    // Initialize outbound_edge_indices_buffer with edge IDs
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(edges_size),
        [outbound_edge_indices_buffer, is_directed] DEVICE (const size_t i) {
            size_t outbound_index = is_directed ? i : i * 2;
            outbound_edge_indices_buffer[outbound_index] = EdgeWithEndpointType{static_cast<long>(i), true};

            if (!is_directed) {
                outbound_edge_indices_buffer[outbound_index + 1] = EdgeWithEndpointType
                        {static_cast<long>(i), false};
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each initialize buffer in compute_node_edge_indices_cuda");

    // Initialize inbound_indices for directed graphs
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_indices(node_edge_index->inbound_indices);
        thrust::sequence(
            DEVICE_EXECUTION_POLICY,
            d_inbound_indices,
            d_inbound_indices + static_cast<long>(edges_size)
        );
        CUDA_KERNEL_CHECK("After thrust sequence in compute_node_edge_indices_cuda");
    }

    // Wrap buffer with device pointer for sorting
    const thrust::device_ptr<EdgeWithEndpointType> d_buffer(outbound_edge_indices_buffer);

    auto sources = edge_data->sources;
    auto targets = edge_data->targets;

    // Sort outbound_edge_indices_buffer by node ID
    thrust::stable_sort(
        DEVICE_EXECUTION_POLICY,
        d_buffer,
        d_buffer + static_cast<long>(buffer_size),
        [sources, targets] DEVICE (
    const EdgeWithEndpointType &a, const EdgeWithEndpointType &b) {
            const int node_a = a.is_source ? sources[a.edge_id] : targets[a.edge_id];
            const int node_b = b.is_source ? sources[b.edge_id] : targets[b.edge_id];
            return node_a < node_b;
        }
    );
    CUDA_KERNEL_CHECK("After thrust stable_sort outbound in compute_node_edge_indices_cuda");

    // Sort inbound_indices for directed graphs
    if (is_directed) {
        const thrust::device_ptr<size_t> d_inbound_indices(node_edge_index->inbound_indices);
        thrust::stable_sort(
            DEVICE_EXECUTION_POLICY,
            d_inbound_indices,
            d_inbound_indices + static_cast<long>(edges_size),
            [targets] DEVICE (const size_t a, const size_t b) {
                return targets[a] < targets[b];
            }
        );
        CUDA_KERNEL_CHECK("After thrust stable_sort inbound in compute_node_edge_indices_cuda");
    }

    // Extract edge_id from buffer to outbound_indices
    const thrust::device_ptr<size_t> d_outbound_indices(node_edge_index->outbound_indices);
    thrust::transform(
        DEVICE_EXECUTION_POLICY,
        d_buffer,
        d_buffer + static_cast<long>(buffer_size),
        d_outbound_indices,
        [] DEVICE (const EdgeWithEndpointType &edge_with_type) {
            return edge_with_type.edge_id;
        }
    );
    CUDA_KERNEL_CHECK("After thrust transform outbound in compute_node_edge_indices_cuda");
}

HOST void node_edge_index::compute_node_timestamp_offsets_cuda(
    NodeEdgeIndexStore *node_edge_index,
    const EdgeDataStore *edge_data,
    const size_t node_count,
    bool is_directed
) {
    // Allocate device memory for temporary arrays to count groups per node
    size_t *d_outbound_group_count = nullptr;
    size_t *d_inbound_group_count = nullptr;

    CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_group_count, node_count * sizeof(size_t)));
    CUDA_CHECK_AND_CLEAR(hipMemset(d_outbound_group_count, 0, node_count * sizeof(size_t)));

    if (is_directed) {
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_group_count, node_count * sizeof(size_t)));
        CUDA_CHECK_AND_CLEAR(hipMemset(d_inbound_group_count, 0, node_count * sizeof(size_t)));
    }

    // Get raw pointers for data access in kernel
    int64_t *timestamps_ptr = edge_data->timestamps;
    size_t *outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t *inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    size_t *outbound_indices_ptr = node_edge_index->outbound_indices;
    size_t *inbound_indices_ptr = is_directed ? node_edge_index->inbound_indices : nullptr;

    size_t *outbound_group_count_ptr = d_outbound_group_count;
    size_t *inbound_group_count_ptr = d_inbound_group_count;

    // Fill timestamp groups counts
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(node_count),
        [outbound_offsets_ptr, inbound_offsets_ptr,
            outbound_indices_ptr, inbound_indices_ptr,
            outbound_group_count_ptr, inbound_group_count_ptr,
            timestamps_ptr, is_directed] DEVICE (const size_t node) {
            // Outbound groups
            size_t start = outbound_offsets_ptr[node];
            size_t end = outbound_offsets_ptr[node + 1];

            if (start < end) {
                outbound_group_count_ptr[node] = 1; // First group always exists

                for (size_t i = start + 1; i < end; ++i) {
                    if (timestamps_ptr[outbound_indices_ptr[i]] !=
                        timestamps_ptr[outbound_indices_ptr[i - 1]]) {
                        atomicAdd(reinterpret_cast<unsigned int *>(&outbound_group_count_ptr[node]), 1);
                    }
                }
            }

            // Inbound groups for directed graphs
            if (is_directed) {
                start = inbound_offsets_ptr[node];
                end = inbound_offsets_ptr[node + 1];

                if (start < end) {
                    inbound_group_count_ptr[node] = 1; // First group always exists

                    for (size_t i = start + 1; i < end; ++i) {
                        if (timestamps_ptr[inbound_indices_ptr[i]] !=
                            timestamps_ptr[inbound_indices_ptr[i - 1]]) {
                            atomicAdd(reinterpret_cast<unsigned int *>(&inbound_group_count_ptr[node]), 1);
                        }
                    }
                }
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each fill groups in compute_node_timestamp_offsets_cuda");

    // Create device pointers for prefix scan
    thrust::device_ptr<size_t> d_outbound_group_count_thrust(d_outbound_group_count);
    thrust::device_ptr<size_t>
            d_outbound_timestamp_group_offsets(node_edge_index->outbound_timestamp_group_offsets);

    // First element should be 0
    CUDA_CHECK_AND_CLEAR(hipMemset(node_edge_index->outbound_timestamp_group_offsets, 0, sizeof(size_t)));

    // Calculate prefix sum for outbound group offsets
    thrust::inclusive_scan(
        DEVICE_EXECUTION_POLICY,
        d_outbound_group_count_thrust,
        d_outbound_group_count_thrust + static_cast<long>(node_count),
        d_outbound_timestamp_group_offsets + 1
    );
    CUDA_KERNEL_CHECK("After thrust inclusive_scan outbound in compute_node_timestamp_offsets_cuda");

    // Inbound processing for directed graphs
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_group_count_thrust(d_inbound_group_count);
        thrust::device_ptr<size_t> d_inbound_timestamp_group_offsets(
            node_edge_index->inbound_timestamp_group_offsets);

        // First element should be 0
        CUDA_CHECK_AND_CLEAR(hipMemset(node_edge_index->inbound_timestamp_group_offsets, 0, sizeof(size_t)));

        // Calculate prefix sum for inbound group offsets
        thrust::inclusive_scan(
            DEVICE_EXECUTION_POLICY,
            d_inbound_group_count_thrust,
            d_inbound_group_count_thrust + static_cast<long>(node_count),
            d_inbound_timestamp_group_offsets + 1
        );
        CUDA_KERNEL_CHECK("After thrust inclusive_scan inbound in compute_node_timestamp_offsets_cuda");
    }

    // Free temporary memory
    CUDA_CHECK_AND_CLEAR(hipFree(d_outbound_group_count));
    if (is_directed) {
        CUDA_CHECK_AND_CLEAR(hipFree(d_inbound_group_count));
    }
}

HOST void node_edge_index::compute_node_timestamp_indices_cuda(
    NodeEdgeIndexStore *node_edge_index,
    const EdgeDataStore *edge_data,
    const size_t node_index_capacity,
    bool is_directed
) {
    // Get raw pointers for data access in kernel
    int64_t *timestamps_ptr = edge_data->timestamps;
    size_t *outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t *inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    size_t *outbound_indices_ptr = node_edge_index->outbound_indices;
    size_t *inbound_indices_ptr = is_directed ? node_edge_index->inbound_indices : nullptr;
    size_t *outbound_timestamp_group_indices_ptr = node_edge_index->outbound_timestamp_group_indices;
    size_t *inbound_timestamp_group_indices_ptr = is_directed
                                                      ? node_edge_index->inbound_timestamp_group_indices
                                                      : nullptr;
    size_t *outbound_timestamp_group_offsets_ptr = node_edge_index->outbound_timestamp_group_offsets;
    size_t *inbound_timestamp_group_offsets_ptr = is_directed
                                                      ? node_edge_index->inbound_timestamp_group_offsets
                                                      : nullptr;

    // Fill timestamp group indices
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(node_index_capacity),
        [outbound_offsets_ptr, inbound_offsets_ptr,
            outbound_indices_ptr, inbound_indices_ptr,
            outbound_timestamp_group_offsets_ptr, inbound_timestamp_group_offsets_ptr,
            outbound_timestamp_group_indices_ptr, inbound_timestamp_group_indices_ptr,
            timestamps_ptr, is_directed] DEVICE (const size_t node) {
            // Outbound timestamp groups
            size_t start = outbound_offsets_ptr[node];
            size_t end = outbound_offsets_ptr[node + 1];
            size_t group_pos = outbound_timestamp_group_offsets_ptr[node];

            if (start < end) {
                // First group always starts at the first edge
                outbound_timestamp_group_indices_ptr[group_pos++] = start;

                for (size_t i = start + 1; i < end; ++i) {
                    if (timestamps_ptr[outbound_indices_ptr[i]] !=
                        timestamps_ptr[outbound_indices_ptr[i - 1]]) {
                        outbound_timestamp_group_indices_ptr[group_pos++] = i;
                    }
                }
            }

            // Inbound timestamp groups for directed graphs
            if (is_directed) {
                start = inbound_offsets_ptr[node];
                end = inbound_offsets_ptr[node + 1];
                group_pos = inbound_timestamp_group_offsets_ptr[node];

                if (start < end) {
                    // First group always starts at the first edge
                    inbound_timestamp_group_indices_ptr[group_pos++] = start;

                    for (size_t i = start + 1; i < end; ++i) {
                        if (timestamps_ptr[inbound_indices_ptr[i]] !=
                            timestamps_ptr[inbound_indices_ptr[i - 1]]) {
                            inbound_timestamp_group_indices_ptr[group_pos++] = i;
                        }
                    }
                }
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each fill indices in compute_node_timestamp_indices_cuda");
}

HOST void node_edge_index::update_temporal_weights_cuda(
    NodeEdgeIndexStore *node_edge_index,
    const EdgeDataStore *edge_data,
    double timescale_bound
) {
    // Get the number of nodes and timestamp groups
    const size_t node_index_capacity = node_edge_index->outbound_offsets_size - 1;
    const size_t outbound_groups_size = node_edge_index->outbound_timestamp_group_indices_size;

    // Resize outbound weight arrays
    resize_memory(
        &node_edge_index->outbound_forward_cumulative_weights_exponential,
        node_edge_index->outbound_forward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu);
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = outbound_groups_size;

    resize_memory(
        &node_edge_index->outbound_backward_cumulative_weights_exponential,
        node_edge_index->outbound_backward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = outbound_groups_size;

    // Resize inbound weights array if directed graph
    if (node_edge_index->inbound_offsets_size > 0) {
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;
        resize_memory(
            &node_edge_index->inbound_backward_cumulative_weights_exponential,
            node_edge_index->inbound_backward_cumulative_weights_exponential_size,
            inbound_groups_size,
            node_edge_index->use_gpu);
        node_edge_index->inbound_backward_cumulative_weights_exponential_size = inbound_groups_size;
    }

    // Process outbound weights
    {
        // Get outbound timestamp group offsets
        MemoryView<size_t> outbound_offsets = get_timestamp_offset_vector(node_edge_index, true, false);

        // Allocate temporary device memory for weights
        double *d_forward_weights = nullptr;
        double *d_backward_weights = nullptr;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_forward_weights, outbound_groups_size * sizeof(double)));
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_backward_weights, outbound_groups_size * sizeof(double)));

        // Get raw pointers for device code
        int64_t *timestamps_ptr = edge_data->timestamps;
        size_t *outbound_indices_ptr = node_edge_index->outbound_indices;
        size_t *outbound_group_indices_ptr = node_edge_index->outbound_timestamp_group_indices;
        size_t *outbound_offsets_ptr = outbound_offsets.data;

        // Calculate weights in parallel for each node
        thrust::for_each(
            DEVICE_EXECUTION_POLICY,
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(node_index_capacity),
            [timestamps_ptr, outbound_indices_ptr, outbound_group_indices_ptr,
                outbound_offsets_ptr, d_forward_weights, d_backward_weights, timescale_bound]
            DEVICE (const size_t node) {
                const size_t out_start = outbound_offsets_ptr[node];
                const size_t out_end = outbound_offsets_ptr[node + 1];

                if (out_start < out_end) {
                    // Get node's timestamp range
                    const size_t first_group_start = outbound_group_indices_ptr[out_start];
                    const size_t last_group_start = outbound_group_indices_ptr[out_end - 1];
                    const int64_t min_ts = timestamps_ptr[outbound_indices_ptr[first_group_start]];
                    const int64_t max_ts = timestamps_ptr[outbound_indices_ptr[last_group_start]];

                    const auto time_diff = static_cast<double>(max_ts - min_ts);
                    const double time_scale = (timescale_bound > 0 && time_diff > 0)
                                                  ? timescale_bound / time_diff
                                                  : 1.0;

                    double forward_sum = 0.0;
                    double backward_sum = 0.0;

                    // Calculate weights for each group
                    for (size_t pos = out_start; pos < out_end; ++pos) {
                        const size_t edge_start = outbound_group_indices_ptr[pos];
                        const int64_t group_ts = timestamps_ptr[outbound_indices_ptr[edge_start]];

                        const auto time_diff_forward = static_cast<double>(max_ts - group_ts);
                        const auto time_diff_backward = static_cast<double>(group_ts - min_ts);

                        const double forward_scaled = timescale_bound > 0
                                                          ? time_diff_forward * time_scale
                                                          : time_diff_forward;
                        const double backward_scaled = timescale_bound > 0
                                                           ? time_diff_backward * time_scale
                                                           : time_diff_backward;

                        const double forward_weight = exp(forward_scaled);
                        d_forward_weights[pos] = forward_weight;
                        forward_sum += forward_weight;

                        const double backward_weight = exp(backward_scaled);
                        d_backward_weights[pos] = backward_weight;
                        backward_sum += backward_weight;
                    }

                    // Normalize and compute cumulative sums
                    double forward_cumsum = 0.0, backward_cumsum = 0.0;
                    for (size_t pos = out_start; pos < out_end; ++pos) {
                        d_forward_weights[pos] /= forward_sum;
                        d_backward_weights[pos] /= backward_sum;

                        forward_cumsum += d_forward_weights[pos];
                        backward_cumsum += d_backward_weights[pos];

                        d_forward_weights[pos] = forward_cumsum;
                        d_backward_weights[pos] = backward_cumsum;
                    }
                }
            }
        );
        CUDA_KERNEL_CHECK("After thrust for_each outbound weights in update_temporal_weights_cuda");

        // Copy results to destination arrays
        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->outbound_forward_cumulative_weights_exponential,
            d_forward_weights,
            outbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->outbound_backward_cumulative_weights_exponential,
            d_backward_weights,
            outbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        // Clean up temporary memory
        CUDA_CHECK_AND_CLEAR(hipFree(d_forward_weights));
        CUDA_CHECK_AND_CLEAR(hipFree(d_backward_weights));
    }

    // Process inbound weights if directed
    if (node_edge_index->inbound_offsets_size > 0) {
        // Get inbound timestamp group offsets
        MemoryView<size_t> inbound_offsets = get_timestamp_offset_vector(node_edge_index, false, true);
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;

        // Allocate temporary device memory for weights
        double *d_backward_weights = nullptr;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_backward_weights, inbound_groups_size * sizeof(double)));

        // Get raw pointers for device code
        int64_t *timestamps_ptr = edge_data->timestamps;
        size_t *inbound_indices_ptr = node_edge_index->inbound_indices;
        size_t *inbound_group_indices_ptr = node_edge_index->inbound_timestamp_group_indices;
        size_t *inbound_offsets_ptr = inbound_offsets.data;

        // Calculate weights in parallel for each node
        thrust::for_each(
            DEVICE_EXECUTION_POLICY,
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(node_index_capacity),
            [timestamps_ptr, inbound_indices_ptr, inbound_group_indices_ptr,
                inbound_offsets_ptr, d_backward_weights, timescale_bound]
            DEVICE (const size_t node) {
                const size_t in_start = inbound_offsets_ptr[node];
                const size_t in_end = inbound_offsets_ptr[node + 1];

                if (in_start < in_end) {
                    // Get node's timestamp range
                    const size_t first_group_start = inbound_group_indices_ptr[in_start];
                    const size_t last_group_start = inbound_group_indices_ptr[in_end - 1];
                    const int64_t min_ts = timestamps_ptr[inbound_indices_ptr[first_group_start]];
                    const int64_t max_ts = timestamps_ptr[inbound_indices_ptr[last_group_start]];

                    const auto time_diff = static_cast<double>(max_ts - min_ts);
                    const double time_scale = (timescale_bound > 0 && time_diff > 0)
                                                  ? timescale_bound / time_diff
                                                  : 1.0;

                    double backward_sum = 0.0;

                    // Calculate weights and sum
                    for (size_t pos = in_start; pos < in_end; ++pos) {
                        const size_t edge_start = inbound_group_indices_ptr[pos];
                        const int64_t group_ts = timestamps_ptr[inbound_indices_ptr[edge_start]];

                        const auto time_diff_backward = static_cast<double>(group_ts - min_ts);
                        const double backward_scaled = timescale_bound > 0
                                                           ? time_diff_backward * time_scale
                                                           : time_diff_backward;

                        const double backward_weight = exp(backward_scaled);
                        d_backward_weights[pos] = backward_weight;
                        backward_sum += backward_weight;
                    }

                    // Normalize and compute cumulative sum
                    double backward_cumsum = 0.0;
                    for (size_t pos = in_start; pos < in_end; ++pos) {
                        d_backward_weights[pos] /= backward_sum;
                        backward_cumsum += d_backward_weights[pos];
                        d_backward_weights[pos] = backward_cumsum;
                    }
                }
            }
        );
        CUDA_KERNEL_CHECK("After thrust for_each inbound weights in update_temporal_weights_cuda");

        // Copy results to destination array
        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->inbound_backward_cumulative_weights_exponential,
            d_backward_weights,
            inbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        // Clean up temporary memory
        CUDA_CHECK_AND_CLEAR(hipFree(d_backward_weights));
    }
}

HOST NodeEdgeIndexStore* node_edge_index::to_device_ptr(const NodeEdgeIndexStore *node_edge_index) {
    // Create a new NodeEdgeIndex object on the device
    NodeEdgeIndexStore *device_node_edge_index;
    CUDA_CHECK_AND_CLEAR(hipMalloc(&device_node_edge_index, sizeof(NodeEdgeIndexStore)));

    // Create a temporary copy to modify for device pointers
    NodeEdgeIndexStore temp_node_edge_index = *node_edge_index;
    temp_node_edge_index.owns_data = false;

    // If already using GPU, just copy the struct with its pointers
    if (!node_edge_index->use_gpu) {
        temp_node_edge_index.owns_data = true;

        // Copy each array to device if it exists
        if (node_edge_index->outbound_offsets) {
            size_t *d_outbound_offsets;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_offsets, node_edge_index->outbound_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_offsets, node_edge_index->outbound_offsets, node_edge_index->
                    outbound_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_offsets = d_outbound_offsets;
        }

        if (node_edge_index->inbound_offsets) {
            size_t *d_inbound_offsets;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_inbound_offsets, node_edge_index->inbound_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_inbound_offsets, node_edge_index->inbound_offsets, node_edge_index->
                    inbound_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_offsets = d_inbound_offsets;
        }

        if (node_edge_index->outbound_indices) {
            size_t *d_outbound_indices;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_indices, node_edge_index->outbound_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_indices, node_edge_index->outbound_indices, node_edge_index->
                    outbound_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_indices = d_outbound_indices;
        }

        if (node_edge_index->inbound_indices) {
            size_t *d_inbound_indices;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_inbound_indices, node_edge_index->inbound_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_inbound_indices, node_edge_index->inbound_indices, node_edge_index->
                    inbound_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_indices = d_inbound_indices;
        }

        if (node_edge_index->outbound_timestamp_group_offsets) {
            size_t *d_outbound_timestamp_group_offsets;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_timestamp_group_offsets, node_edge_index->
                    outbound_timestamp_group_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_timestamp_group_offsets, node_edge_index->outbound_timestamp_group_offsets,
                    node_edge_index->outbound_timestamp_group_offsets_size * sizeof(size_t), hipMemcpyHostToDevice
                ));
            temp_node_edge_index.outbound_timestamp_group_offsets = d_outbound_timestamp_group_offsets;
        }

        if (node_edge_index->inbound_timestamp_group_offsets) {
            size_t *d_inbound_timestamp_group_offsets;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_inbound_timestamp_group_offsets, node_edge_index->inbound_timestamp_group_offsets_size
                    * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_inbound_timestamp_group_offsets, node_edge_index->inbound_timestamp_group_offsets,
                    node_edge_index->inbound_timestamp_group_offsets_size * sizeof(size_t), hipMemcpyHostToDevice))
            ;
            temp_node_edge_index.inbound_timestamp_group_offsets = d_inbound_timestamp_group_offsets;
        }

        if (node_edge_index->outbound_timestamp_group_indices) {
            size_t *d_outbound_timestamp_group_indices;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_timestamp_group_indices, node_edge_index->
                    outbound_timestamp_group_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_timestamp_group_indices, node_edge_index->outbound_timestamp_group_indices,
                    node_edge_index->outbound_timestamp_group_indices_size * sizeof(size_t), hipMemcpyHostToDevice
                ));
            temp_node_edge_index.outbound_timestamp_group_indices = d_outbound_timestamp_group_indices;
        }

        if (node_edge_index->inbound_timestamp_group_indices) {
            size_t *d_inbound_timestamp_group_indices;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_inbound_timestamp_group_indices, node_edge_index->inbound_timestamp_group_indices_size
                    * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_inbound_timestamp_group_indices, node_edge_index->inbound_timestamp_group_indices,
                    node_edge_index->inbound_timestamp_group_indices_size * sizeof(size_t), hipMemcpyHostToDevice))
            ;
            temp_node_edge_index.inbound_timestamp_group_indices = d_inbound_timestamp_group_indices;
        }

        if (node_edge_index->outbound_forward_cumulative_weights_exponential) {
            double *d_outbound_forward_weights;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_forward_weights, node_edge_index->
                    outbound_forward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_forward_weights, node_edge_index->
                    outbound_forward_cumulative_weights_exponential,
                    node_edge_index->outbound_forward_cumulative_weights_exponential_size * sizeof(double),
                    hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_forward_cumulative_weights_exponential = d_outbound_forward_weights;
        }

        if (node_edge_index->outbound_backward_cumulative_weights_exponential) {
            double *d_outbound_backward_weights;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_outbound_backward_weights, node_edge_index->
                    outbound_backward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_outbound_backward_weights, node_edge_index->
                    outbound_backward_cumulative_weights_exponential,
                    node_edge_index->outbound_backward_cumulative_weights_exponential_size * sizeof(double),
                    hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_backward_cumulative_weights_exponential = d_outbound_backward_weights;
        }

        if (node_edge_index->inbound_backward_cumulative_weights_exponential) {
            double *d_inbound_backward_weights;
            CUDA_CHECK_AND_CLEAR(
                hipMalloc(&d_inbound_backward_weights, node_edge_index->
                    inbound_backward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(
                hipMemcpy(d_inbound_backward_weights, node_edge_index->
                    inbound_backward_cumulative_weights_exponential,
                    node_edge_index->inbound_backward_cumulative_weights_exponential_size * sizeof(double),
                    hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_backward_cumulative_weights_exponential = d_inbound_backward_weights;
        }

        // Make sure use_gpu is set to true
        temp_node_edge_index.use_gpu = true;
    }

    CUDA_CHECK_AND_CLEAR(
        hipMemcpy(device_node_edge_index, &temp_node_edge_index, sizeof(NodeEdgeIndexStore), hipMemcpyHostToDevice
        ));

    temp_node_edge_index.owns_data = false;

    return device_node_edge_index;
}

#endif

HOST void node_edge_index::rebuild(NodeEdgeIndexStore *node_edge_index, const EdgeDataStore *edge_data, const bool is_directed) {
    // Get sizes
    const size_t num_edges = edge_data->timestamps_size;

    // Step 2: Allocate and compute node edge offsets
    allocate_node_edge_offsets(node_edge_index, edge_data->active_node_ids_size, is_directed);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_edge_offsets_cuda(node_edge_index, edge_data, is_directed);
    } else
    #endif
    {
        compute_node_edge_offsets_std(node_edge_index, edge_data, is_directed);
    }

    // Step 3: Allocate and compute node edge indices
    allocate_node_edge_indices(node_edge_index, is_directed);

    // Create buffer for outbound edge indices
    size_t outbound_edge_indices_len = is_directed ? num_edges : num_edges * 2;
    EdgeWithEndpointType *outbound_edge_indices_buffer = nullptr;
    allocate_memory(&outbound_edge_indices_buffer, outbound_edge_indices_len, node_edge_index->use_gpu);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_edge_indices_cuda(node_edge_index, edge_data, outbound_edge_indices_buffer, is_directed);
    } else
    #endif
    {
        compute_node_edge_indices_std(node_edge_index, edge_data, outbound_edge_indices_buffer, is_directed);
    }

    // Clean up edge indices buffer
    clear_memory(&outbound_edge_indices_buffer, node_edge_index->use_gpu);

    // Step 4: Compute node timestamp offsets
    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_timestamp_offsets_cuda(node_edge_index, edge_data, edge_data->active_node_ids_size,
                                            is_directed);
    } else
    #endif
    {
        compute_node_timestamp_offsets_std(node_edge_index, edge_data, edge_data->active_node_ids_size,
                                           is_directed);
    }

    // Step 5: Allocate and compute node timestamp indices
    allocate_node_timestamp_indices(node_edge_index, is_directed);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_timestamp_indices_cuda(node_edge_index, edge_data, edge_data->active_node_ids_size,
                                            is_directed);
    } else
    #endif
    {
        compute_node_timestamp_indices_std(node_edge_index, edge_data, edge_data->active_node_ids_size,
                                           is_directed);
    }
}
