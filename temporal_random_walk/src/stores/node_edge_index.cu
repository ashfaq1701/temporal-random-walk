#include "hip/hip_runtime.h"
#include "node_edge_index.cuh"

#include <cmath>
#include <algorithm>
#include "../common/cuda_config.cuh"
#include "../common/error_handlers.cuh"

#ifdef HAS_CUDA
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#endif

#include "../common/memory.cuh"

HOST void node_edge_index::clear(NodeEdgeIndexStore* node_edge_index) {
    // Clear edge CSR structures
    clear_memory(&node_edge_index->outbound_offsets, node_edge_index->use_gpu);
    node_edge_index->outbound_offsets_size = 0;

    clear_memory(&node_edge_index->outbound_indices, node_edge_index->use_gpu);
    node_edge_index->outbound_indices_size = 0;

    clear_memory(&node_edge_index->outbound_timestamp_group_offsets, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_offsets_size = 0;

    clear_memory(&node_edge_index->outbound_timestamp_group_indices, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_indices_size = 0;

    // Clear inbound structures
    clear_memory(&node_edge_index->inbound_offsets, node_edge_index->use_gpu);
    node_edge_index->inbound_offsets_size = 0;

    clear_memory(&node_edge_index->inbound_indices, node_edge_index->use_gpu);
    node_edge_index->inbound_indices_size = 0;

    clear_memory(&node_edge_index->inbound_timestamp_group_offsets, node_edge_index->use_gpu);
    node_edge_index->inbound_timestamp_group_offsets_size = 0;

    clear_memory(&node_edge_index->inbound_timestamp_group_indices, node_edge_index->use_gpu);
    node_edge_index->inbound_timestamp_group_indices_size = 0;

    // Clear temporal weights
    clear_memory(&node_edge_index->outbound_forward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = 0;

    clear_memory(&node_edge_index->outbound_backward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = 0;

    clear_memory(&node_edge_index->inbound_backward_cumulative_weights_exponential, node_edge_index->use_gpu);
    node_edge_index->inbound_backward_cumulative_weights_exponential_size = 0;
}

HOST DEVICE SizeRange node_edge_index::get_edge_range(const NodeEdgeIndexStore* node_edge_index, const int dense_node_id, const bool forward, const bool is_directed) {
    if (is_directed) {
        const size_t* offsets = forward ? node_edge_index->outbound_offsets : node_edge_index->inbound_offsets;
        size_t offsets_size = forward ? node_edge_index->outbound_offsets_size : node_edge_index->inbound_offsets_size;

        if (dense_node_id < 0 || dense_node_id >= offsets_size - 1) {
            return SizeRange{0, 0};
        }

        const size_t start = offsets[dense_node_id];
        const size_t end = offsets[dense_node_id + 1];

        return SizeRange{start, end};
    } else {
        if (dense_node_id < 0 || dense_node_id >= node_edge_index->outbound_offsets_size - 1) {
            return SizeRange{0, 0};
        }

        const size_t start = node_edge_index->outbound_offsets[dense_node_id];
        const size_t end = node_edge_index->outbound_offsets[dense_node_id + 1];

        return SizeRange{start, end};
    }
}

HOST DEVICE SizeRange node_edge_index::get_timestamp_group_range(const NodeEdgeIndexStore* node_edge_index, const int dense_node_id, const size_t group_idx, const bool forward, const bool is_directed) {
    const size_t* group_offsets = nullptr;
    size_t group_offsets_size = 0;
    const size_t* group_indices = nullptr;
    const size_t* edge_offsets = nullptr;

    if (is_directed && !forward) {
        group_offsets = node_edge_index->inbound_timestamp_group_offsets;
        group_offsets_size = node_edge_index->inbound_timestamp_group_offsets_size;
        group_indices = node_edge_index->inbound_timestamp_group_indices;
        edge_offsets = node_edge_index->inbound_offsets;
    } else {
        group_offsets = node_edge_index->outbound_timestamp_group_offsets;
        group_offsets_size = node_edge_index->outbound_timestamp_group_offsets_size;
        group_indices = node_edge_index->outbound_timestamp_group_indices;
        edge_offsets = node_edge_index->outbound_offsets;
    }

    if (dense_node_id < 0 || dense_node_id >= group_offsets_size - 1) {
        return SizeRange{0, 0};
    }

    const size_t node_group_start = group_offsets[dense_node_id];
    const size_t node_group_end = group_offsets[dense_node_id + 1];

    const size_t num_groups = node_group_end - node_group_start;
    if (group_idx >= num_groups) {
        return SizeRange{0, 0};
    }

    const size_t group_start_idx = node_group_start + group_idx;
    const size_t group_start = group_indices[group_start_idx];

    // Group end is either next group's start or node's edge range end
    size_t group_end = 0;
    if (group_idx == num_groups - 1) {
        group_end = edge_offsets[dense_node_id + 1];
    } else {
        group_end = group_indices[group_start_idx + 1];
    }

    return SizeRange{group_start, group_end};
}

HOST DEVICE size_t node_edge_index::get_timestamp_group_count(const NodeEdgeIndexStore* node_edge_index, const int dense_node_id, const bool forward, const bool is_directed) {
    // Get the appropriate timestamp offset vector
    MemoryView<size_t> offsets_block = get_timestamp_offset_vector(node_edge_index, forward, is_directed);
    const size_t* offsets = offsets_block.data;
    size_t offsets_size = offsets_block.size;

    // Check if the node ID is valid
    if (dense_node_id < 0 || dense_node_id >= offsets_size - 1) {
        return 0;
    }

    // Get start and end offsets for the node
    const size_t start = offsets[dense_node_id];
    const size_t end = offsets[dense_node_id + 1];

    return end - start;
}

HOST DEVICE MemoryView<size_t> node_edge_index::get_timestamp_offset_vector(const NodeEdgeIndexStore* node_edge_index, const bool forward, const bool is_directed) {
    if (is_directed && !forward) {
        return MemoryView<size_t>{
            node_edge_index->inbound_timestamp_group_offsets,
            node_edge_index->inbound_timestamp_group_offsets_size
        };
    } else {
        return MemoryView<size_t>{
            node_edge_index->outbound_timestamp_group_offsets,
            node_edge_index->outbound_timestamp_group_offsets_size
        };
    }
}

HOST void node_edge_index::allocate_node_edge_offsets(NodeEdgeIndexStore* node_edge_index, size_t node_index_capacity, bool is_directed) {
    allocate_memory(&node_edge_index->outbound_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
    node_edge_index->outbound_offsets_size = node_index_capacity + 1;
    fill_memory(node_edge_index->outbound_offsets, node_index_capacity + 1, static_cast<size_t>(0), node_edge_index->use_gpu);

    allocate_memory(&node_edge_index->outbound_timestamp_group_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_offsets_size = node_index_capacity + 1;
    fill_memory(node_edge_index->outbound_timestamp_group_offsets, node_index_capacity + 1, static_cast<size_t>(0), node_edge_index->use_gpu);

    // For directed graphs, also allocate inbound structures
    if (is_directed) {
        allocate_memory(&node_edge_index->inbound_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
        node_edge_index->inbound_offsets_size = node_index_capacity + 1;
        fill_memory(node_edge_index->inbound_offsets, node_index_capacity + 1, static_cast<size_t>(0), node_edge_index->use_gpu);

        allocate_memory(&node_edge_index->inbound_timestamp_group_offsets, node_index_capacity + 1, node_edge_index->use_gpu);
        node_edge_index->inbound_timestamp_group_offsets_size = node_index_capacity + 1;
        fill_memory(node_edge_index->inbound_timestamp_group_offsets, node_index_capacity + 1, static_cast<size_t>(0), node_edge_index->use_gpu);
    }
}

HOST void node_edge_index::allocate_node_edge_indices(NodeEdgeIndexStore* node_edge_index, bool is_directed) {
    size_t num_outbound_edges = 0;

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        // For GPU memory, we need to copy the value back to host
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_outbound_edges,
                  node_edge_index->outbound_offsets + (node_edge_index->outbound_offsets_size - 1),
                  sizeof(size_t),
                  hipMemcpyDeviceToHost));
    }
    else
    #endif
    {
        // For CPU memory, we can access it directly
        num_outbound_edges = node_edge_index->outbound_offsets[node_edge_index->outbound_offsets_size - 1];
    }

    // Allocate memory for outbound indices
    allocate_memory(&node_edge_index->outbound_indices, num_outbound_edges, node_edge_index->use_gpu);
    node_edge_index->outbound_indices_size = num_outbound_edges;

    // For directed graphs, also allocate inbound indices
    if (is_directed) {
        size_t num_inbound_edges = 0;

        #ifdef HAS_CUDA
        if (node_edge_index->use_gpu) {
            CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_inbound_edges,
                      node_edge_index->inbound_offsets + (node_edge_index->inbound_offsets_size - 1),
                      sizeof(size_t),
                      hipMemcpyDeviceToHost));
        }
        else
        #endif
        {
            num_inbound_edges = node_edge_index->inbound_offsets[node_edge_index->inbound_offsets_size - 1];
        }

        allocate_memory(&node_edge_index->inbound_indices, num_inbound_edges, node_edge_index->use_gpu);
        node_edge_index->inbound_indices_size = num_inbound_edges;
    }
}

HOST void node_edge_index::allocate_node_timestamp_indices(NodeEdgeIndexStore* node_edge_index, bool is_directed) {
    size_t num_outbound_groups = 0;

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        // For GPU memory, we need to copy the value back to host
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_outbound_groups,
                  node_edge_index->outbound_timestamp_group_offsets + (node_edge_index->outbound_timestamp_group_offsets_size - 1),
                  sizeof(size_t),
                  hipMemcpyDeviceToHost));
    }
    else
    #endif
    {
        // For CPU memory, we can access it directly
        num_outbound_groups = node_edge_index->outbound_timestamp_group_offsets[node_edge_index->outbound_timestamp_group_offsets_size - 1];
    }

    // Allocate memory for outbound timestamp group indices
    allocate_memory(&node_edge_index->outbound_timestamp_group_indices, num_outbound_groups, node_edge_index->use_gpu);
    node_edge_index->outbound_timestamp_group_indices_size = num_outbound_groups;

    // For directed graphs, also allocate inbound timestamp group indices
    if (is_directed) {
        size_t num_inbound_groups = 0;

        #ifdef HAS_CUDA
        if (node_edge_index->use_gpu) {
            CUDA_CHECK_AND_CLEAR(hipMemcpy(&num_inbound_groups,
                      node_edge_index->inbound_timestamp_group_offsets + (node_edge_index->inbound_timestamp_group_offsets_size - 1),
                      sizeof(size_t),
                      hipMemcpyDeviceToHost));
        }
        else
        #endif
        {
            num_inbound_groups = node_edge_index->inbound_timestamp_group_offsets[node_edge_index->inbound_timestamp_group_offsets_size - 1];
        }

        allocate_memory(&node_edge_index->inbound_timestamp_group_indices, num_inbound_groups, node_edge_index->use_gpu);
        node_edge_index->inbound_timestamp_group_indices_size = num_inbound_groups;
    }
}

HOST void node_edge_index::populate_dense_ids_std(
    EdgeDataStore* edge_data,
    NodeMappingStore* node_mapping,
    int* dense_sources,
    int* dense_targets
) {
    // Iterate through all edges
    for (size_t i = 0; i < edge_data->timestamps_size; i++) {
        // Convert sparse IDs to dense IDs using node mapping
        const int sparse_src = edge_data->sources[i];
        const int sparse_tgt = edge_data->targets[i];

        // Use the to_dense function from the node_mapping namespace
        dense_sources[i] = node_mapping::to_dense(node_mapping, sparse_src);
        dense_targets[i] = node_mapping::to_dense(node_mapping, sparse_tgt);
    }
}

HOST void node_edge_index::compute_node_edge_offsets_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const int* dense_sources,
    const int* dense_targets,
    const bool is_directed
) {
    // First pass: count edges per node
    for (size_t i = 0; i < edge_data->timestamps_size; i++) {
        const int src_idx = dense_sources[i];
        const int tgt_idx = dense_targets[i];

        // Count outbound edges (increment the count at index src_idx + 1)
        node_edge_index->outbound_offsets[src_idx + 1]++;

        if (is_directed) {
            // For directed graphs, also increment inbound edges count
            node_edge_index->inbound_offsets[tgt_idx + 1]++;
        } else {
            // For undirected graphs, each edge appears in both directions
            node_edge_index->outbound_offsets[tgt_idx + 1]++;
        }
    }

    // Calculate prefix sums for edge offsets
    for (size_t i = 1; i < node_edge_index->outbound_offsets_size; i++) {
        node_edge_index->outbound_offsets[i] += node_edge_index->outbound_offsets[i-1];

        if (is_directed) {
            node_edge_index->inbound_offsets[i] += node_edge_index->inbound_offsets[i-1];
        }
    }
}

HOST void node_edge_index::compute_node_edge_indices_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const int* dense_sources,
    const int* dense_targets,
    EdgeWithEndpointType* outbound_edge_indices_buffer,
    const bool is_directed
) {
    const size_t edges_size = edge_data->timestamps_size;

    // Fill the buffer with edge information
    for (size_t i = 0; i < edges_size; i++) {
        const size_t outbound_index = is_directed ? i : i * 2;
        outbound_edge_indices_buffer[outbound_index] = EdgeWithEndpointType{static_cast<long>(i), true};

        if (is_directed) {
            // For directed graphs, simply assign each edge ID to inbound_indices
            node_edge_index->inbound_indices[i] = i;
        } else {
            // For undirected graphs, add each edge in both directions
            outbound_edge_indices_buffer[outbound_index + 1] = EdgeWithEndpointType{static_cast<long>(i), false};
        }
    }

    const size_t buffer_size = is_directed ? edges_size : edges_size * 2;

    // Sort outbound edge indices by node ID
    std::stable_sort(
        outbound_edge_indices_buffer,
        outbound_edge_indices_buffer + buffer_size,
        [dense_sources, dense_targets](const EdgeWithEndpointType& a, const EdgeWithEndpointType& b) {
            const int node_a = a.is_source ? dense_sources[a.edge_id] : dense_targets[a.edge_id];
            const int node_b = b.is_source ? dense_sources[b.edge_id] : dense_targets[b.edge_id];
            return node_a < node_b;
        }
    );

    // Sort inbound indices for directed graphs by target node
    if (is_directed) {
        std::stable_sort(
            node_edge_index->inbound_indices,
            node_edge_index->inbound_indices + edges_size,
            [dense_targets](size_t a, size_t b) {
                return dense_targets[a] < dense_targets[b];
            }
        );
    }

    // Extract edge IDs from buffer to outbound_indices
    for (size_t i = 0; i < buffer_size; i++) {
        node_edge_index->outbound_indices[i] = outbound_edge_indices_buffer[i].edge_id;
    }
}

HOST void node_edge_index::compute_node_timestamp_offsets_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_index_capacity,
    const bool is_directed
) {
    // Temporary arrays to store group counts for each node
    auto* outbound_group_count = new size_t[node_index_capacity]();  // Initialize to zeros
    size_t* inbound_group_count = nullptr;

    if (is_directed) {
        inbound_group_count = new size_t[node_index_capacity]();  // Initialize to zeros
    }

    // Count timestamp groups for each node
    for (size_t node = 0; node < node_index_capacity; node++) {
        // Process outbound groups
        size_t start = node_edge_index->outbound_offsets[node];
        size_t end = node_edge_index->outbound_offsets[node + 1];

        if (start < end) {
            outbound_group_count[node] = 1;  // First group always exists if there are edges

            for (size_t i = start + 1; i < end; ++i) {
                size_t curr_edge_id = node_edge_index->outbound_indices[i];
                size_t prev_edge_id = node_edge_index->outbound_indices[i-1];

                if (edge_data->timestamps[curr_edge_id] != edge_data->timestamps[prev_edge_id]) {
                    ++outbound_group_count[node];  // New timestamp group
                }
            }
        }

        // Process inbound groups for directed graphs
        if (is_directed) {
            start = node_edge_index->inbound_offsets[node];
            end = node_edge_index->inbound_offsets[node + 1];

            if (start < end) {
                inbound_group_count[node] = 1;  // First group always exists if there are edges

                for (size_t i = start + 1; i < end; ++i) {
                    size_t curr_edge_id = node_edge_index->inbound_indices[i];
                    size_t prev_edge_id = node_edge_index->inbound_indices[i-1];

                    if (edge_data->timestamps[curr_edge_id] != edge_data->timestamps[prev_edge_id]) {
                        ++inbound_group_count[node];  // New timestamp group
                    }
                }
            }
        }
    }

    // Calculate prefix sums for group offsets
    node_edge_index->outbound_timestamp_group_offsets[0] = 0;  // Start at 0

    for (size_t i = 0; i < node_index_capacity; i++) {
        node_edge_index->outbound_timestamp_group_offsets[i + 1] =
            node_edge_index->outbound_timestamp_group_offsets[i] + outbound_group_count[i];

        if (is_directed) {
            node_edge_index->inbound_timestamp_group_offsets[i + 1] =
                node_edge_index->inbound_timestamp_group_offsets[i] + inbound_group_count[i];
        }
    }

    // Clean up temporary arrays
    delete[] outbound_group_count;
    delete[] inbound_group_count;
}

HOST void node_edge_index::compute_node_timestamp_indices_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_index_capacity,
    const bool is_directed
) {
    // Process each node
    for (size_t node = 0; node < node_index_capacity; node++) {
        // Fill outbound timestamp group indices
        size_t start = node_edge_index->outbound_offsets[node];
        size_t end = node_edge_index->outbound_offsets[node + 1];
        size_t group_pos = node_edge_index->outbound_timestamp_group_offsets[node];

        if (start < end) {
            // First group always starts at the first edge
            node_edge_index->outbound_timestamp_group_indices[group_pos++] = start;

            for (size_t i = start + 1; i < end; ++i) {
                size_t curr_edge_id = node_edge_index->outbound_indices[i];
                size_t prev_edge_id = node_edge_index->outbound_indices[i-1];

                if (edge_data->timestamps[curr_edge_id] != edge_data->timestamps[prev_edge_id]) {
                    // New group starts at current position
                    node_edge_index->outbound_timestamp_group_indices[group_pos++] = i;
                }
            }
        }

        // Fill inbound timestamp group indices for directed graphs
        if (is_directed) {
            start = node_edge_index->inbound_offsets[node];
            end = node_edge_index->inbound_offsets[node + 1];
            group_pos = node_edge_index->inbound_timestamp_group_offsets[node];

            if (start < end) {
                // First group always starts at the first edge
                node_edge_index->inbound_timestamp_group_indices[group_pos++] = start;

                for (size_t i = start + 1; i < end; ++i) {
                    size_t curr_edge_id = node_edge_index->inbound_indices[i];
                    size_t prev_edge_id = node_edge_index->inbound_indices[i-1];

                    if (edge_data->timestamps[curr_edge_id] != edge_data->timestamps[prev_edge_id]) {
                        // New group starts at current position
                        node_edge_index->inbound_timestamp_group_indices[group_pos++] = i;
                    }
                }
            }
        }
    }
}

HOST void node_edge_index::update_temporal_weights_std(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const double timescale_bound
) {
    const size_t node_index_capacity = node_edge_index->outbound_offsets_size - 1;

    // Resize temporal weights arrays
    const size_t outbound_groups_size = node_edge_index->outbound_timestamp_group_indices_size;

    // Allocate or resize outbound weights arrays
    resize_memory(
        &node_edge_index->outbound_forward_cumulative_weights_exponential,
        node_edge_index->outbound_forward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = outbound_groups_size;

    resize_memory(
        &node_edge_index->outbound_backward_cumulative_weights_exponential,
        node_edge_index->outbound_backward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = outbound_groups_size;

    // Allocate inbound weights for directed graphs
    if (node_edge_index->inbound_offsets_size > 0) {
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;
        resize_memory(
            &node_edge_index->inbound_backward_cumulative_weights_exponential,
            node_edge_index->inbound_backward_cumulative_weights_exponential_size,
            inbound_groups_size,
            node_edge_index->use_gpu
        );
        node_edge_index->inbound_backward_cumulative_weights_exponential_size = inbound_groups_size;
    }

    // Process each node
    for (size_t node = 0; node < node_index_capacity; node++) {
        // Outbound weights
        MemoryView<size_t> outbound_offsets = get_timestamp_offset_vector(node_edge_index, true, false);
        const size_t out_start = outbound_offsets.data[node];
        const size_t out_end = outbound_offsets.data[node + 1];

        if (out_start < out_end) {
            const size_t first_group_start = node_edge_index->outbound_timestamp_group_indices[out_start];
            const size_t last_group_start = node_edge_index->outbound_timestamp_group_indices[out_end - 1];
            const int64_t min_ts = edge_data->timestamps[node_edge_index->outbound_indices[first_group_start]];
            const int64_t max_ts = edge_data->timestamps[node_edge_index->outbound_indices[last_group_start]];
            const auto time_diff = static_cast<double>(max_ts - min_ts);
            const double time_scale = (timescale_bound > 0 && time_diff > 0) ?
                timescale_bound / time_diff : 1.0;

            double forward_sum = 0.0;
            double backward_sum = 0.0;

            // Calculate weights and sums
            for (size_t pos = out_start; pos < out_end; ++pos) {
                const size_t edge_start = node_edge_index->outbound_timestamp_group_indices[pos];
                const int64_t group_ts = edge_data->timestamps[node_edge_index->outbound_indices[edge_start]];

                const auto time_diff_forward = static_cast<double>(max_ts - group_ts);
                const auto time_diff_backward = static_cast<double>(group_ts - min_ts);

                const double forward_scaled = timescale_bound > 0 ?
                    time_diff_forward * time_scale : time_diff_forward;
                const double backward_scaled = timescale_bound > 0 ?
                    time_diff_backward * time_scale : time_diff_backward;

                const double forward_weight = exp(forward_scaled);
                node_edge_index->outbound_forward_cumulative_weights_exponential[pos] = forward_weight;
                forward_sum += forward_weight;

                const double backward_weight = exp(backward_scaled);
                node_edge_index->outbound_backward_cumulative_weights_exponential[pos] = backward_weight;
                backward_sum += backward_weight;
            }

            // Normalize and compute cumulative sums
            double forward_cumsum = 0.0, backward_cumsum = 0.0;
            for (size_t pos = out_start; pos < out_end; ++pos) {
                node_edge_index->outbound_forward_cumulative_weights_exponential[pos] /= forward_sum;
                node_edge_index->outbound_backward_cumulative_weights_exponential[pos] /= backward_sum;

                forward_cumsum += node_edge_index->outbound_forward_cumulative_weights_exponential[pos];
                backward_cumsum += node_edge_index->outbound_backward_cumulative_weights_exponential[pos];

                node_edge_index->outbound_forward_cumulative_weights_exponential[pos] = forward_cumsum;
                node_edge_index->outbound_backward_cumulative_weights_exponential[pos] = backward_cumsum;
            }
        }

        // Inbound weights
        if (node_edge_index->inbound_offsets_size > 0) {
            MemoryView<size_t> inbound_offsets = get_timestamp_offset_vector(node_edge_index, false, true);
            const size_t in_start = inbound_offsets.data[node];
            const size_t in_end = inbound_offsets.data[node + 1];

            if (in_start < in_end) {
                const size_t first_group_start = node_edge_index->inbound_timestamp_group_indices[in_start];
                const size_t last_group_start = node_edge_index->inbound_timestamp_group_indices[in_end - 1];
                const int64_t min_ts = edge_data->timestamps[node_edge_index->inbound_indices[first_group_start]];
                const int64_t max_ts = edge_data->timestamps[node_edge_index->inbound_indices[last_group_start]];
                const auto time_diff = static_cast<double>(max_ts - min_ts);
                const double time_scale = (timescale_bound > 0 && time_diff > 0) ?
                    timescale_bound / time_diff : 1.0;

                double backward_sum = 0.0;

                // Calculate weights and sum
                for (size_t pos = in_start; pos < in_end; ++pos) {
                    const size_t edge_start = node_edge_index->inbound_timestamp_group_indices[pos];
                    const int64_t group_ts = edge_data->timestamps[node_edge_index->inbound_indices[edge_start]];

                    const auto time_diff_backward = static_cast<double>(group_ts - min_ts);
                    const double backward_scaled = timescale_bound > 0 ?
                        time_diff_backward * time_scale : time_diff_backward;

                    const double backward_weight = exp(backward_scaled);
                    node_edge_index->inbound_backward_cumulative_weights_exponential[pos] = backward_weight;
                    backward_sum += backward_weight;
                }

                // Normalize and compute cumulative sum
                double backward_cumsum = 0.0;
                for (size_t pos = in_start; pos < in_end; ++pos) {
                    node_edge_index->inbound_backward_cumulative_weights_exponential[pos] /= backward_sum;
                    backward_cumsum += node_edge_index->inbound_backward_cumulative_weights_exponential[pos];
                    node_edge_index->inbound_backward_cumulative_weights_exponential[pos] = backward_cumsum;
                }
            }
        }
    }
}

#ifdef HAS_CUDA

HOST void node_edge_index::populate_dense_ids_cuda(
    const EdgeDataStore* edge_data,
    const NodeMappingStore* node_mapping,
    int* dense_sources,
    int* dense_targets
) {
    // Create device pointers from raw pointers
    thrust::device_ptr<int> d_sources(edge_data->sources);
    thrust::device_ptr<int> d_targets(edge_data->targets);
    thrust::device_ptr<int> d_dense_sources(dense_sources);
    thrust::device_ptr<int> d_dense_targets(dense_targets);

    // Get raw pointer to sparse_to_dense mapping
    int* node_index_ptr = node_mapping->node_index;
    size_t node_index_capacity = node_mapping->capacity;

    // Transform source IDs from sparse to dense
    thrust::transform(
        DEVICE_EXECUTION_POLICY,
        d_sources,
        d_sources + static_cast<long>(edge_data->sources_size),
        d_dense_sources,
        [node_index_ptr, node_index_capacity] DEVICE (const int id) {
            return node_mapping::to_dense_from_ptr_device(node_index_ptr, id, static_cast<int>(node_index_capacity));
        }
    );
    CUDA_KERNEL_CHECK("After thrust transform sources in populate_dense_ids_cuda");

    // Transform target IDs from sparse to dense
    thrust::transform(
        DEVICE_EXECUTION_POLICY,
        d_targets,
        d_targets + static_cast<long>(edge_data->targets_size),
        d_dense_targets,
        [node_index_ptr, node_index_capacity] DEVICE (const int id) {
            return node_mapping::to_dense_from_ptr_device(node_index_ptr, id, static_cast<int>(node_index_capacity));
        }
    );
    CUDA_KERNEL_CHECK("After thrust transform targets in populate_dense_ids_cuda");
}

HOST void node_edge_index::compute_node_edge_offsets_cuda(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    int* dense_sources,
    int* dense_targets,
    bool is_directed
) {
    const size_t num_edges = edge_data->timestamps_size;

    // Get raw pointers to work with
    size_t* outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t* inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    int* src_ptr = dense_sources;
    int* tgt_ptr = dense_targets;

    // Count edges per node using atomics
    auto counter_device_lambda = [
        outbound_offsets_ptr, inbound_offsets_ptr,
        src_ptr, tgt_ptr, is_directed] DEVICE (const size_t i) {
        const int src_idx = src_ptr[i];
        const int tgt_idx = tgt_ptr[i];

        atomicAdd(reinterpret_cast<unsigned int *>(&outbound_offsets_ptr[src_idx + 1]), 1);
        if (is_directed) {
            atomicAdd(reinterpret_cast<unsigned int *>(&inbound_offsets_ptr[tgt_idx + 1]), 1);
        } else {
            atomicAdd(reinterpret_cast<unsigned int *>(&outbound_offsets_ptr[tgt_idx + 1]), 1);
        }
    };

    // Process all edges in parallel
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(num_edges),
        counter_device_lambda);
    CUDA_KERNEL_CHECK("After thrust for_each in compute_node_edge_offsets_cuda");

    // Calculate prefix sums for outbound edge offsets
    thrust::device_ptr<size_t> d_outbound_offsets(outbound_offsets_ptr);
    thrust::inclusive_scan(
        DEVICE_EXECUTION_POLICY,
        d_outbound_offsets + 1,
        d_outbound_offsets + static_cast<long>(node_edge_index->outbound_offsets_size),
        d_outbound_offsets + 1
    );
    CUDA_KERNEL_CHECK("After thrust inclusive_scan outbound in compute_node_edge_offsets_cuda");

    // Calculate prefix sums for inbound edge offsets (if directed)
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_offsets(inbound_offsets_ptr);
        thrust::inclusive_scan(
            DEVICE_EXECUTION_POLICY,
            d_inbound_offsets + 1,
            d_inbound_offsets + static_cast<long>(node_edge_index->inbound_offsets_size),
            d_inbound_offsets + 1
        );
        CUDA_KERNEL_CHECK("After thrust inclusive_scan inbound in compute_node_edge_offsets_cuda");
    }
}

HOST void node_edge_index::compute_node_edge_indices_cuda(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const int* dense_sources,
    const int* dense_targets,
    EdgeWithEndpointType* outbound_edge_indices_buffer,
    bool is_directed) {

    const size_t edges_size = edge_data->timestamps_size;
    const size_t buffer_size = is_directed ? edges_size : edges_size * 2;

    // Initialize outbound_edge_indices_buffer with edge IDs
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(edges_size),
        [outbound_edge_indices_buffer, is_directed] DEVICE (const size_t i) {
            size_t outbound_index = is_directed ? i : i * 2;
            outbound_edge_indices_buffer[outbound_index] = EdgeWithEndpointType{static_cast<long>(i), true};

            if (!is_directed) {
                outbound_edge_indices_buffer[outbound_index + 1] = EdgeWithEndpointType{static_cast<long>(i), false};
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each initialize buffer in compute_node_edge_indices_cuda");

    // Initialize inbound_indices for directed graphs
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_indices(node_edge_index->inbound_indices);
        thrust::sequence(
            DEVICE_EXECUTION_POLICY,
            d_inbound_indices,
            d_inbound_indices + static_cast<long>(edges_size)
        );
        CUDA_KERNEL_CHECK("After thrust sequence in compute_node_edge_indices_cuda");
    }

    // Wrap buffer with device pointer for sorting
    thrust::device_ptr<EdgeWithEndpointType> d_buffer(outbound_edge_indices_buffer);

    // Sort outbound_edge_indices_buffer by node ID
    thrust::stable_sort(
        DEVICE_EXECUTION_POLICY,
        d_buffer,
        d_buffer + static_cast<long>(buffer_size),
        [dense_sources, dense_targets] DEVICE (
            const EdgeWithEndpointType& a, const EdgeWithEndpointType& b) {
            const int node_a = a.is_source ? dense_sources[a.edge_id] : dense_targets[a.edge_id];
            const int node_b = b.is_source ? dense_sources[b.edge_id] : dense_targets[b.edge_id];
            return node_a < node_b;
        }
    );
    CUDA_KERNEL_CHECK("After thrust stable_sort outbound in compute_node_edge_indices_cuda");

    // Sort inbound_indices for directed graphs
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_indices(node_edge_index->inbound_indices);
        thrust::stable_sort(
            DEVICE_EXECUTION_POLICY,
            d_inbound_indices,
            d_inbound_indices + static_cast<long>(edges_size),
            [dense_targets] DEVICE (size_t a, size_t b) {
                return dense_targets[a] < dense_targets[b];
            }
        );
        CUDA_KERNEL_CHECK("After thrust stable_sort inbound in compute_node_edge_indices_cuda");
    }

    // Extract edge_id from buffer to outbound_indices
    thrust::device_ptr<size_t> d_outbound_indices(node_edge_index->outbound_indices);
    thrust::transform(
        DEVICE_EXECUTION_POLICY,
        d_buffer,
        d_buffer + static_cast<long>(buffer_size),
        d_outbound_indices,
        [] DEVICE (const EdgeWithEndpointType& edge_with_type) {
            return edge_with_type.edge_id;
        }
    );
    CUDA_KERNEL_CHECK("After thrust transform outbound in compute_node_edge_indices_cuda");
}

HOST void node_edge_index::compute_node_timestamp_offsets_cuda(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_index_capacity,
    const bool is_directed
) {
    // Allocate device memory for temporary arrays to count groups per node
    size_t* d_outbound_group_count = nullptr;
    size_t* d_inbound_group_count = nullptr;

    CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_group_count, node_index_capacity * sizeof(size_t)));
    CUDA_CHECK_AND_CLEAR(hipMemset(d_outbound_group_count, 0, node_index_capacity * sizeof(size_t)));

    if (is_directed) {
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_group_count, node_index_capacity * sizeof(size_t)));
        CUDA_CHECK_AND_CLEAR(hipMemset(d_inbound_group_count, 0, node_index_capacity * sizeof(size_t)));
    }

    // Get raw pointers for data access in kernel
    int64_t* timestamps_ptr = edge_data->timestamps;
    size_t* outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t* inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    size_t* outbound_indices_ptr = node_edge_index->outbound_indices;
    size_t* inbound_indices_ptr = is_directed ? node_edge_index->inbound_indices : nullptr;

    size_t* outbound_group_count_ptr = d_outbound_group_count;
    size_t* inbound_group_count_ptr = d_inbound_group_count;

    // Fill timestamp groups counts
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(node_index_capacity),
        [outbound_offsets_ptr, inbound_offsets_ptr,
         outbound_indices_ptr, inbound_indices_ptr,
         outbound_group_count_ptr, inbound_group_count_ptr,
         timestamps_ptr, is_directed] DEVICE (const size_t node) {
            // Outbound groups
            size_t start = outbound_offsets_ptr[node];
            size_t end = outbound_offsets_ptr[node + 1];

            if (start < end) {
                outbound_group_count_ptr[node] = 1; // First group always exists

                for (size_t i = start + 1; i < end; ++i) {
                    if (timestamps_ptr[outbound_indices_ptr[i]] !=
                        timestamps_ptr[outbound_indices_ptr[i - 1]]) {
                        atomicAdd(reinterpret_cast<unsigned int*>(&outbound_group_count_ptr[node]), 1);
                    }
                }
            }

            // Inbound groups for directed graphs
            if (is_directed) {
                start = inbound_offsets_ptr[node];
                end = inbound_offsets_ptr[node + 1];

                if (start < end) {
                    inbound_group_count_ptr[node] = 1; // First group always exists

                    for (size_t i = start + 1; i < end; ++i) {
                        if (timestamps_ptr[inbound_indices_ptr[i]] !=
                            timestamps_ptr[inbound_indices_ptr[i - 1]]) {
                            atomicAdd(reinterpret_cast<unsigned int*>(&inbound_group_count_ptr[node]), 1);
                        }
                    }
                }
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each fill groups in compute_node_timestamp_offsets_cuda");

    // Create device pointers for prefix scan
    thrust::device_ptr<size_t> d_outbound_group_count_thrust(d_outbound_group_count);
    thrust::device_ptr<size_t> d_outbound_timestamp_group_offsets(node_edge_index->outbound_timestamp_group_offsets);

    // First element should be 0
    CUDA_CHECK_AND_CLEAR(hipMemset(node_edge_index->outbound_timestamp_group_offsets, 0, sizeof(size_t)));

    // Calculate prefix sum for outbound group offsets
    thrust::inclusive_scan(
        DEVICE_EXECUTION_POLICY,
        d_outbound_group_count_thrust,
        d_outbound_group_count_thrust + static_cast<long>(node_index_capacity),
        d_outbound_timestamp_group_offsets + 1
    );
    CUDA_KERNEL_CHECK("After thrust inclusive_scan outbound in compute_node_timestamp_offsets_cuda");

    // Inbound processing for directed graphs
    if (is_directed) {
        thrust::device_ptr<size_t> d_inbound_group_count_thrust(d_inbound_group_count);
        thrust::device_ptr<size_t> d_inbound_timestamp_group_offsets(node_edge_index->inbound_timestamp_group_offsets);

        // First element should be 0
        CUDA_CHECK_AND_CLEAR(hipMemset(node_edge_index->inbound_timestamp_group_offsets, 0, sizeof(size_t)));

        // Calculate prefix sum for inbound group offsets
        thrust::inclusive_scan(
            DEVICE_EXECUTION_POLICY,
            d_inbound_group_count_thrust,
            d_inbound_group_count_thrust + static_cast<long>(node_index_capacity),
            d_inbound_timestamp_group_offsets + 1
        );
        CUDA_KERNEL_CHECK("After thrust inclusive_scan inbound in compute_node_timestamp_offsets_cuda");
    }

    // Free temporary memory
    CUDA_CHECK_AND_CLEAR(hipFree(d_outbound_group_count));
    if (is_directed) {
        CUDA_CHECK_AND_CLEAR(hipFree(d_inbound_group_count));
    }
}

HOST void node_edge_index::compute_node_timestamp_indices_cuda(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    const size_t node_index_capacity,
    const bool is_directed
) {
    // Get raw pointers for data access in kernel
    int64_t* timestamps_ptr = edge_data->timestamps;
    size_t* outbound_offsets_ptr = node_edge_index->outbound_offsets;
    size_t* inbound_offsets_ptr = is_directed ? node_edge_index->inbound_offsets : nullptr;
    size_t* outbound_indices_ptr = node_edge_index->outbound_indices;
    size_t* inbound_indices_ptr = is_directed ? node_edge_index->inbound_indices : nullptr;
    size_t* outbound_timestamp_group_indices_ptr = node_edge_index->outbound_timestamp_group_indices;
    size_t* inbound_timestamp_group_indices_ptr = is_directed ? node_edge_index->inbound_timestamp_group_indices : nullptr;
    size_t* outbound_timestamp_group_offsets_ptr = node_edge_index->outbound_timestamp_group_offsets;
    size_t* inbound_timestamp_group_offsets_ptr = is_directed ? node_edge_index->inbound_timestamp_group_offsets : nullptr;

    // Fill timestamp group indices
    thrust::for_each(
        DEVICE_EXECUTION_POLICY,
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(node_index_capacity),
        [outbound_offsets_ptr, inbound_offsets_ptr,
         outbound_indices_ptr, inbound_indices_ptr,
         outbound_timestamp_group_offsets_ptr, inbound_timestamp_group_offsets_ptr,
         outbound_timestamp_group_indices_ptr, inbound_timestamp_group_indices_ptr,
         timestamps_ptr, is_directed] DEVICE (const size_t node) {
            // Outbound timestamp groups
            size_t start = outbound_offsets_ptr[node];
            size_t end = outbound_offsets_ptr[node + 1];
            size_t group_pos = outbound_timestamp_group_offsets_ptr[node];

            if (start < end) {
                // First group always starts at the first edge
                outbound_timestamp_group_indices_ptr[group_pos++] = start;

                for (size_t i = start + 1; i < end; ++i) {
                    if (timestamps_ptr[outbound_indices_ptr[i]] !=
                        timestamps_ptr[outbound_indices_ptr[i-1]]) {
                        outbound_timestamp_group_indices_ptr[group_pos++] = i;
                    }
                }
            }

            // Inbound timestamp groups for directed graphs
            if (is_directed) {
                start = inbound_offsets_ptr[node];
                end = inbound_offsets_ptr[node + 1];
                group_pos = inbound_timestamp_group_offsets_ptr[node];

                if (start < end) {
                    // First group always starts at the first edge
                    inbound_timestamp_group_indices_ptr[group_pos++] = start;

                    for (size_t i = start + 1; i < end; ++i) {
                        if (timestamps_ptr[inbound_indices_ptr[i]] !=
                            timestamps_ptr[inbound_indices_ptr[i-1]]) {
                            inbound_timestamp_group_indices_ptr[group_pos++] = i;
                        }
                    }
                }
            }
        }
    );
    CUDA_KERNEL_CHECK("After thrust for_each fill indices in compute_node_timestamp_indices_cuda");
}

HOST void node_edge_index::update_temporal_weights_cuda(
    NodeEdgeIndexStore* node_edge_index,
    const EdgeDataStore* edge_data,
    double timescale_bound
) {
    // Get the number of nodes and timestamp groups
    const size_t node_index_capacity = node_edge_index->outbound_offsets_size - 1;
    const size_t outbound_groups_size = node_edge_index->outbound_timestamp_group_indices_size;

    // Resize outbound weight arrays
    resize_memory(
        &node_edge_index->outbound_forward_cumulative_weights_exponential,
        node_edge_index->outbound_forward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu);
    node_edge_index->outbound_forward_cumulative_weights_exponential_size = outbound_groups_size;

    resize_memory(
        &node_edge_index->outbound_backward_cumulative_weights_exponential,
        node_edge_index->outbound_backward_cumulative_weights_exponential_size,
        outbound_groups_size,
        node_edge_index->use_gpu
    );
    node_edge_index->outbound_backward_cumulative_weights_exponential_size = outbound_groups_size;

    // Resize inbound weights array if directed graph
    if (node_edge_index->inbound_offsets_size > 0) {
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;
        resize_memory(
            &node_edge_index->inbound_backward_cumulative_weights_exponential,
            node_edge_index->inbound_backward_cumulative_weights_exponential_size,
            inbound_groups_size,
            node_edge_index->use_gpu);
        node_edge_index->inbound_backward_cumulative_weights_exponential_size = inbound_groups_size;
    }

    // Process outbound weights
    {
        // Get outbound timestamp group offsets
        MemoryView<size_t> outbound_offsets = get_timestamp_offset_vector(node_edge_index, true, false);

        // Allocate temporary device memory for weights
        double* d_forward_weights = nullptr;
        double* d_backward_weights = nullptr;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_forward_weights, outbound_groups_size * sizeof(double)));
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_backward_weights, outbound_groups_size * sizeof(double)));

        // Get raw pointers for device code
        int64_t* timestamps_ptr = edge_data->timestamps;
        size_t* outbound_indices_ptr = node_edge_index->outbound_indices;
        size_t* outbound_group_indices_ptr = node_edge_index->outbound_timestamp_group_indices;
        size_t* outbound_offsets_ptr = outbound_offsets.data;

        // Calculate weights in parallel for each node
        thrust::for_each(
            DEVICE_EXECUTION_POLICY,
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(node_index_capacity),
            [timestamps_ptr, outbound_indices_ptr, outbound_group_indices_ptr,
             outbound_offsets_ptr, d_forward_weights, d_backward_weights, timescale_bound]
             DEVICE (const size_t node) {
                const size_t out_start = outbound_offsets_ptr[node];
                const size_t out_end = outbound_offsets_ptr[node + 1];

                if (out_start < out_end) {
                    // Get node's timestamp range
                    const size_t first_group_start = outbound_group_indices_ptr[out_start];
                    const size_t last_group_start = outbound_group_indices_ptr[out_end - 1];
                    const int64_t min_ts = timestamps_ptr[outbound_indices_ptr[first_group_start]];
                    const int64_t max_ts = timestamps_ptr[outbound_indices_ptr[last_group_start]];

                    const auto time_diff = static_cast<double>(max_ts - min_ts);
                    const double time_scale = (timescale_bound > 0 && time_diff > 0) ?
                        timescale_bound / time_diff : 1.0;

                    double forward_sum = 0.0;
                    double backward_sum = 0.0;

                    // Calculate weights for each group
                    for (size_t pos = out_start; pos < out_end; ++pos) {
                        const size_t edge_start = outbound_group_indices_ptr[pos];
                        const int64_t group_ts = timestamps_ptr[outbound_indices_ptr[edge_start]];

                        const auto time_diff_forward = static_cast<double>(max_ts - group_ts);
                        const auto time_diff_backward = static_cast<double>(group_ts - min_ts);

                        const double forward_scaled = timescale_bound > 0 ?
                            time_diff_forward * time_scale : time_diff_forward;
                        const double backward_scaled = timescale_bound > 0 ?
                            time_diff_backward * time_scale : time_diff_backward;

                        const double forward_weight = exp(forward_scaled);
                        d_forward_weights[pos] = forward_weight;
                        forward_sum += forward_weight;

                        const double backward_weight = exp(backward_scaled);
                        d_backward_weights[pos] = backward_weight;
                        backward_sum += backward_weight;
                    }

                    // Normalize and compute cumulative sums
                    double forward_cumsum = 0.0, backward_cumsum = 0.0;
                    for (size_t pos = out_start; pos < out_end; ++pos) {
                        d_forward_weights[pos] /= forward_sum;
                        d_backward_weights[pos] /= backward_sum;

                        forward_cumsum += d_forward_weights[pos];
                        backward_cumsum += d_backward_weights[pos];

                        d_forward_weights[pos] = forward_cumsum;
                        d_backward_weights[pos] = backward_cumsum;
                    }
                }
            }
        );
        CUDA_KERNEL_CHECK("After thrust for_each outbound weights in update_temporal_weights_cuda");

        // Copy results to destination arrays
        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->outbound_forward_cumulative_weights_exponential,
            d_forward_weights,
            outbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->outbound_backward_cumulative_weights_exponential,
            d_backward_weights,
            outbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        // Clean up temporary memory
        CUDA_CHECK_AND_CLEAR(hipFree(d_forward_weights));
        CUDA_CHECK_AND_CLEAR(hipFree(d_backward_weights));
    }

    // Process inbound weights if directed
    if (node_edge_index->inbound_offsets_size > 0) {
        // Get inbound timestamp group offsets
        MemoryView<size_t> inbound_offsets = get_timestamp_offset_vector(node_edge_index, false, true);
        const size_t inbound_groups_size = node_edge_index->inbound_timestamp_group_indices_size;

        // Allocate temporary device memory for weights
        double* d_backward_weights = nullptr;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_backward_weights, inbound_groups_size * sizeof(double)));

        // Get raw pointers for device code
        int64_t* timestamps_ptr = edge_data->timestamps;
        size_t* inbound_indices_ptr = node_edge_index->inbound_indices;
        size_t* inbound_group_indices_ptr = node_edge_index->inbound_timestamp_group_indices;
        size_t* inbound_offsets_ptr = inbound_offsets.data;

        // Calculate weights in parallel for each node
        thrust::for_each(
            DEVICE_EXECUTION_POLICY,
            thrust::make_counting_iterator<size_t>(0),
            thrust::make_counting_iterator<size_t>(node_index_capacity),
            [timestamps_ptr, inbound_indices_ptr, inbound_group_indices_ptr,
             inbound_offsets_ptr, d_backward_weights, timescale_bound]
             DEVICE (const size_t node) {
                const size_t in_start = inbound_offsets_ptr[node];
                const size_t in_end = inbound_offsets_ptr[node + 1];

                if (in_start < in_end) {
                    // Get node's timestamp range
                    const size_t first_group_start = inbound_group_indices_ptr[in_start];
                    const size_t last_group_start = inbound_group_indices_ptr[in_end - 1];
                    const int64_t min_ts = timestamps_ptr[inbound_indices_ptr[first_group_start]];
                    const int64_t max_ts = timestamps_ptr[inbound_indices_ptr[last_group_start]];

                    const auto time_diff = static_cast<double>(max_ts - min_ts);
                    const double time_scale = (timescale_bound > 0 && time_diff > 0) ?
                        timescale_bound / time_diff : 1.0;

                    double backward_sum = 0.0;

                    // Calculate weights and sum
                    for (size_t pos = in_start; pos < in_end; ++pos) {
                        const size_t edge_start = inbound_group_indices_ptr[pos];
                        const int64_t group_ts = timestamps_ptr[inbound_indices_ptr[edge_start]];

                        const auto time_diff_backward = static_cast<double>(group_ts - min_ts);
                        const double backward_scaled = timescale_bound > 0 ?
                            time_diff_backward * time_scale : time_diff_backward;

                        const double backward_weight = exp(backward_scaled);
                        d_backward_weights[pos] = backward_weight;
                        backward_sum += backward_weight;
                    }

                    // Normalize and compute cumulative sum
                    double backward_cumsum = 0.0;
                    for (size_t pos = in_start; pos < in_end; ++pos) {
                        d_backward_weights[pos] /= backward_sum;
                        backward_cumsum += d_backward_weights[pos];
                        d_backward_weights[pos] = backward_cumsum;
                    }
                }
            }
        );
        CUDA_KERNEL_CHECK("After thrust for_each inbound weights in update_temporal_weights_cuda");

        // Copy results to destination array
        CUDA_CHECK_AND_CLEAR(hipMemcpy(
            node_edge_index->inbound_backward_cumulative_weights_exponential,
            d_backward_weights,
            inbound_groups_size * sizeof(double),
            hipMemcpyDeviceToDevice
        ));

        // Clean up temporary memory
        CUDA_CHECK_AND_CLEAR(hipFree(d_backward_weights));
    }
}

HOST NodeEdgeIndexStore* node_edge_index::to_device_ptr(const NodeEdgeIndexStore* node_edge_index) {
    // Create a new NodeEdgeIndex object on the device
    NodeEdgeIndexStore* device_node_edge_index;
    CUDA_CHECK_AND_CLEAR(hipMalloc(&device_node_edge_index, sizeof(NodeEdgeIndexStore)));

    // Create a temporary copy to modify for device pointers
    NodeEdgeIndexStore temp_node_edge_index = *node_edge_index;
    temp_node_edge_index.owns_data = false;

    // If already using GPU, just copy the struct with its pointers
    if (node_edge_index->use_gpu) {
        CUDA_CHECK_AND_CLEAR(hipMemcpy(device_node_edge_index, node_edge_index, sizeof(NodeEdgeIndexStore), hipMemcpyHostToDevice));
    } else {
        temp_node_edge_index.owns_data = true;

        // Copy each array to device if it exists
        if (node_edge_index->outbound_offsets) {
            size_t* d_outbound_offsets;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_offsets, node_edge_index->outbound_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_offsets, node_edge_index->outbound_offsets, node_edge_index->outbound_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_offsets = d_outbound_offsets;
        }

        if (node_edge_index->inbound_offsets) {
            size_t* d_inbound_offsets;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_offsets, node_edge_index->inbound_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_inbound_offsets, node_edge_index->inbound_offsets, node_edge_index->inbound_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_offsets = d_inbound_offsets;
        }

        if (node_edge_index->outbound_indices) {
            size_t* d_outbound_indices;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_indices, node_edge_index->outbound_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_indices, node_edge_index->outbound_indices, node_edge_index->outbound_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_indices = d_outbound_indices;
        }

        if (node_edge_index->inbound_indices) {
            size_t* d_inbound_indices;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_indices, node_edge_index->inbound_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_inbound_indices, node_edge_index->inbound_indices, node_edge_index->inbound_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_indices = d_inbound_indices;
        }

        if (node_edge_index->outbound_timestamp_group_offsets) {
            size_t* d_outbound_timestamp_group_offsets;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_timestamp_group_offsets, node_edge_index->outbound_timestamp_group_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_timestamp_group_offsets, node_edge_index->outbound_timestamp_group_offsets,
                      node_edge_index->outbound_timestamp_group_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_timestamp_group_offsets = d_outbound_timestamp_group_offsets;
        }

        if (node_edge_index->inbound_timestamp_group_offsets) {
            size_t* d_inbound_timestamp_group_offsets;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_timestamp_group_offsets, node_edge_index->inbound_timestamp_group_offsets_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_inbound_timestamp_group_offsets, node_edge_index->inbound_timestamp_group_offsets,
                      node_edge_index->inbound_timestamp_group_offsets_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_timestamp_group_offsets = d_inbound_timestamp_group_offsets;
        }

        if (node_edge_index->outbound_timestamp_group_indices) {
            size_t* d_outbound_timestamp_group_indices;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_timestamp_group_indices, node_edge_index->outbound_timestamp_group_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_timestamp_group_indices, node_edge_index->outbound_timestamp_group_indices,
                      node_edge_index->outbound_timestamp_group_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_timestamp_group_indices = d_outbound_timestamp_group_indices;
        }

        if (node_edge_index->inbound_timestamp_group_indices) {
            size_t* d_inbound_timestamp_group_indices;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_timestamp_group_indices, node_edge_index->inbound_timestamp_group_indices_size * sizeof(size_t)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_inbound_timestamp_group_indices, node_edge_index->inbound_timestamp_group_indices,
                      node_edge_index->inbound_timestamp_group_indices_size * sizeof(size_t), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_timestamp_group_indices = d_inbound_timestamp_group_indices;
        }

        if (node_edge_index->outbound_forward_cumulative_weights_exponential) {
            double* d_outbound_forward_weights;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_forward_weights, node_edge_index->outbound_forward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_forward_weights, node_edge_index->outbound_forward_cumulative_weights_exponential,
                      node_edge_index->outbound_forward_cumulative_weights_exponential_size * sizeof(double), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_forward_cumulative_weights_exponential = d_outbound_forward_weights;
        }

        if (node_edge_index->outbound_backward_cumulative_weights_exponential) {
            double* d_outbound_backward_weights;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_outbound_backward_weights, node_edge_index->outbound_backward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_outbound_backward_weights, node_edge_index->outbound_backward_cumulative_weights_exponential,
                      node_edge_index->outbound_backward_cumulative_weights_exponential_size * sizeof(double), hipMemcpyHostToDevice));
            temp_node_edge_index.outbound_backward_cumulative_weights_exponential = d_outbound_backward_weights;
        }

        if (node_edge_index->inbound_backward_cumulative_weights_exponential) {
            double* d_inbound_backward_weights;
            CUDA_CHECK_AND_CLEAR(hipMalloc(&d_inbound_backward_weights, node_edge_index->inbound_backward_cumulative_weights_exponential_size * sizeof(double)));
            CUDA_CHECK_AND_CLEAR(hipMemcpy(d_inbound_backward_weights, node_edge_index->inbound_backward_cumulative_weights_exponential,
                      node_edge_index->inbound_backward_cumulative_weights_exponential_size * sizeof(double), hipMemcpyHostToDevice));
            temp_node_edge_index.inbound_backward_cumulative_weights_exponential = d_inbound_backward_weights;
        }

        // Make sure use_gpu is set to true
        temp_node_edge_index.use_gpu = true;

        // Copy the updated struct to device
        CUDA_CHECK_AND_CLEAR(hipMemcpy(device_node_edge_index, &temp_node_edge_index, sizeof(NodeEdgeIndexStore), hipMemcpyHostToDevice));
    }

    temp_node_edge_index.owns_data = false;

    return device_node_edge_index;
}

#endif

HOST void node_edge_index::rebuild(NodeEdgeIndexStore* node_edge_index, EdgeDataStore* edge_data, NodeMappingStore* node_mapping, bool is_directed) {
    // Get sizes
    const size_t node_index_capacity = node_mapping->capacity;
    const size_t num_edges = edge_data->timestamps_size;

    // Allocate buffers for dense IDs
    int* dense_sources = nullptr;
    int* dense_targets = nullptr;
    allocate_memory(&dense_sources, num_edges, node_edge_index->use_gpu);
    allocate_memory(&dense_targets, num_edges, node_edge_index->use_gpu);

    // Step 1: Populate dense IDs
    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        populate_dense_ids_cuda(edge_data, node_mapping, dense_sources, dense_targets);
    }
    else
    #endif
    {
        populate_dense_ids_std(edge_data, node_mapping, dense_sources, dense_targets);
    }

    // Step 2: Allocate and compute node edge offsets
    allocate_node_edge_offsets(node_edge_index, node_index_capacity, is_directed);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_edge_offsets_cuda(node_edge_index, edge_data, dense_sources, dense_targets, is_directed);
    }
    else
    #endif
    {
        compute_node_edge_offsets_std(node_edge_index, edge_data, dense_sources, dense_targets, is_directed);
    }

    // Step 3: Allocate and compute node edge indices
    allocate_node_edge_indices(node_edge_index, is_directed);

    // Create buffer for outbound edge indices
    size_t outbound_edge_indices_len = is_directed ? num_edges : num_edges * 2;
    EdgeWithEndpointType* outbound_edge_indices_buffer = nullptr;
    allocate_memory(&outbound_edge_indices_buffer, outbound_edge_indices_len, node_edge_index->use_gpu);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_edge_indices_cuda(node_edge_index, edge_data, dense_sources, dense_targets, outbound_edge_indices_buffer, is_directed);
    }
    else
    #endif
    {
        compute_node_edge_indices_std(node_edge_index, edge_data, dense_sources, dense_targets, outbound_edge_indices_buffer, is_directed);
    }

    // Clean up edge indices buffer
    clear_memory(&outbound_edge_indices_buffer, node_edge_index->use_gpu);

    // Step 4: Compute node timestamp offsets
    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_timestamp_offsets_cuda(node_edge_index, edge_data, node_index_capacity, is_directed);
    }
    else
    #endif
    {
        compute_node_timestamp_offsets_std(node_edge_index, edge_data, node_index_capacity, is_directed);
    }

    // Step 5: Allocate and compute node timestamp indices
    allocate_node_timestamp_indices(node_edge_index, is_directed);

    #ifdef HAS_CUDA
    if (node_edge_index->use_gpu) {
        compute_node_timestamp_indices_cuda(node_edge_index, edge_data, node_index_capacity, is_directed);
    }
    else
    #endif
    {
        compute_node_timestamp_indices_std(node_edge_index, edge_data, node_index_capacity, is_directed);
    }

    // Clean up dense ID buffers
    clear_memory(&dense_sources, node_edge_index->use_gpu);
    clear_memory(&dense_targets, node_edge_index->use_gpu);
}
