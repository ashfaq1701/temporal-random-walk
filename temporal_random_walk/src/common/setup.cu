#include "hip/hip_runtime.h"
#include "setup.cuh"

#include <random>

#include "cuda_config.cuh"
#include "error_handlers.cuh"

unsigned long get_random_seed() {
    std::random_device rd;
    const unsigned long seed = rd() ^ (time(nullptr) << 1);
    return seed;
}

#ifdef HAS_CUDA

__global__ void setup_curand_states(hiprandState* rand_states, const unsigned long seed) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, tid, 0, &rand_states[tid]);
}

HOST hiprandState* get_cuda_rand_states(size_t grid_dim, size_t block_dim) {
    const size_t total_threads = grid_dim * block_dim;

    hiprandState* rand_states;
    CUDA_CHECK_AND_CLEAR(hipMalloc(&rand_states, total_threads * sizeof(hiprandState)));

    setup_curand_states<<<grid_dim, block_dim>>>(rand_states, get_random_seed());
    CUDA_KERNEL_CHECK("After hiprand states initialization");

    return rand_states;
}

HOST std::pair<size_t, size_t> get_optimal_launch_params(const size_t data_size, const hipDeviceProp_t* device_prop) {
    size_t block_dim = BLOCK_DIM;
    size_t grid_dim = (data_size + block_dim - 1) / block_dim;
    const size_t min_grid_size = 2 * device_prop->multiProcessorCount;
    grid_dim = std::max(grid_dim, min_grid_size);
    return {grid_dim, block_dim};
}

#endif
