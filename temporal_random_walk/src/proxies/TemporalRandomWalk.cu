#include "hip/hip_runtime.h"
#include "TemporalRandomWalk.cuh"

#include <algorithm>
#include <iterator>

#include "../common/error_handlers.cuh"

#ifdef HAS_CUDA

__global__ void get_edge_count_kernel(size_t* result, const TemporalRandomWalkStore* temporal_random_walk) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

#endif

TemporalRandomWalk::TemporalRandomWalk(
        const bool is_directed,
        const bool use_gpu,
        const int64_t max_time_capacity,
        const bool enable_weight_computation,
        const double timescale_bound,
        const size_t n_threads): use_gpu(use_gpu) {
    temporal_random_walk = new TemporalRandomWalkStore(
        is_directed,
        use_gpu,
        max_time_capacity,
        enable_weight_computation,
        timescale_bound,
        n_threads);
}

TemporalRandomWalk::~TemporalRandomWalk() {
    delete temporal_random_walk;
}

void TemporalRandomWalk::add_multiple_edges(const std::vector<std::tuple<int, int, int64_t>>& edges) const {
    const auto edge_array = new Edge[edges.size()];
    for (size_t idx = 0; idx < edges.size(); idx++) {
        const auto& [u, i, ts] = edges[idx];
        edge_array[idx] = Edge(u, i, ts);
    }

    temporal_random_walk::add_multiple_edges(temporal_random_walk, edge_array, edges.size());

    delete[] edge_array;
}

WalkSet TemporalRandomWalk::get_random_walks_and_times_for_all_nodes_raw(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_per_node,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {
    WalkSet walk_set;

    #ifdef HAS_CUDA
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    }
    else
    #endif
    {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    }

    return walk_set;
}

std::vector<std::tuple<std::vector<int>, std::vector<int64_t>>> TemporalRandomWalk::get_random_walks_and_times_for_all_nodes(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_per_node,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    const WalkSet walk_set = get_random_walks_and_times_for_all_nodes_raw(
        max_walk_len,
        walk_bias,
        num_walks_per_node,
        initial_edge_bias,
        walk_direction);

    std::vector<std::tuple<std::vector<int>, std::vector<int64_t>>> result;

    std::mutex result_mutex;

    #pragma omp parallel for default(none) shared(walk_set, max_walk_len, result, result_mutex)
    for (size_t walk_idx = 0; walk_idx < walk_set.num_walks; walk_idx++) {
        const size_t walk_len = walk_set.walk_lens[walk_idx];

        // Skip empty walks
        if (walk_len == 0) continue;

        // Calculate base pointers for this walk
        const int* nodes_ptr = walk_set.nodes + (walk_idx * max_walk_len);
        const int64_t* timestamps_ptr = walk_set.timestamps + (walk_idx * max_walk_len);

        std::vector<int> nodes(nodes_ptr, nodes_ptr + walk_len);
        std::vector<int64_t> timestamps(timestamps_ptr, timestamps_ptr + walk_len);

        std::lock_guard<std::mutex> lock(result_mutex);
        result.emplace_back(std::move(nodes), std::move(timestamps));
    }

    return result;
}

WalkSet TemporalRandomWalk::get_random_walks_and_times_raw(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_total,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    WalkSet walk_set;

    #ifdef HAS_CUDA
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    }
    else
    #endif
    {
        walk_set = temporal_random_walk::get_random_walks_and_times_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    }

    return walk_set;
}

std::vector<std::tuple<std::vector<int>, std::vector<int64_t>>> TemporalRandomWalk::get_random_walks_and_times(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_total,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    const WalkSet walk_set = get_random_walks_and_times_raw(
        max_walk_len,
        walk_bias,
        num_walks_total,
        initial_edge_bias,
        walk_direction);

    std::vector<std::tuple<std::vector<int>, std::vector<int64_t>>> result;

    std::mutex result_mutex;

    #pragma omp parallel for default(none) shared(walk_set, max_walk_len, result, result_mutex)
    for (size_t walk_idx = 0; walk_idx < walk_set.num_walks; walk_idx++) {
        const size_t walk_len = walk_set.walk_lens[walk_idx];

        // Skip empty walks
        if (walk_len == 0) continue;

        // Calculate base pointers for this walk
        const int* nodes_ptr = walk_set.nodes + (walk_idx * max_walk_len);
        const int64_t* timestamps_ptr = walk_set.timestamps + (walk_idx * max_walk_len);

        std::vector<int> nodes(nodes_ptr, nodes_ptr + walk_len);
        std::vector<int64_t> timestamps(timestamps_ptr, timestamps_ptr + walk_len);

        std::lock_guard<std::mutex> lock(result_mutex);
        result.emplace_back(std::move(nodes), std::move(timestamps));
    }

    return result;
}

size_t TemporalRandomWalk::get_node_count() const {
    return temporal_random_walk::get_node_count(temporal_random_walk);
}

size_t TemporalRandomWalk::get_edge_count() const {
    #ifdef HAS_CUDA
    if (use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_result, sizeof(size_t)));

        TemporalRandomWalkStore* d_temporal_random_walk = temporal_random_walk::to_device_ptr(temporal_random_walk);
        get_edge_count_kernel<<<1, 1>>>(d_result, d_temporal_random_walk);
        CUDA_KERNEL_CHECK("After get_edge_count_kernel execution");

        size_t host_result;
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost));

        CUDA_CHECK_AND_CLEAR(hipFree(d_result));
        CUDA_CHECK_AND_CLEAR(hipFree(d_temporal_random_walk));

        return host_result;
    }
    else
    #endif
    {
        // Direct call for CPU implementation
        return temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

std::vector<int> TemporalRandomWalk::get_node_ids() const {
    const DataBlock<int> node_ids = temporal_random_walk::get_node_ids(temporal_random_walk);
    std::vector<int> result;

    #ifdef HAS_CUDA
    if (node_ids.use_gpu) {
        // Allocate temporary host memory
        int* host_data = new int[node_ids.size];
        CUDA_CHECK_AND_CLEAR(hipMemcpy(host_data, node_ids.data,
                                     node_ids.size * sizeof(int),
                                     hipMemcpyDeviceToHost));

        result.assign(host_data, host_data + node_ids.size);

        delete[] host_data;
    }
    else
    #endif
    {
        result.assign(node_ids.data, node_ids.data + node_ids.size);
    }

    return result;
}

std::vector<std::tuple<int, int, int64_t>> TemporalRandomWalk::get_edges() {
    const DataBlock<Edge> edges = temporal_random_walk::get_edges(temporal_random_walk);
    std::vector<std::tuple<int, int, int64_t>> result;
    result.reserve(edges.size);

    #ifdef HAS_CUDA
    if (edges.use_gpu) {
        auto host_edges = new Edge[edges.size];
        CUDA_CHECK_AND_CLEAR(hipMemcpy(host_edges, edges.data,
                                    edges.size * sizeof(Edge),
                                    hipMemcpyDeviceToHost));

        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(
                host_edges[i].u,
                host_edges[i].i,
                host_edges[i].ts);
        }

        delete[] host_edges;
    }
    else
    #endif
    {
        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(
                edges.data[i].u,
                edges.data[i].i,
                edges.data[i].ts);
        }
    }

    return result;
}

bool TemporalRandomWalk::get_is_directed() const {
    return temporal_random_walk::get_is_directed(temporal_random_walk);
}

void TemporalRandomWalk::clear() const {
    temporal_random_walk::clear(temporal_random_walk);
}
