#include "hip/hip_runtime.h"
#include "TemporalRandomWalk.cuh"

#include <algorithm>
#include <iterator>

#include "../common/error_handlers.cuh"

#ifdef HAS_CUDA

__global__ void get_edge_count_kernel(size_t* result, const TemporalRandomWalkStore* temporal_random_walk) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

#endif

TemporalRandomWalk::TemporalRandomWalk(
        const bool is_directed,
        const bool use_gpu,
        const int64_t max_time_capacity,
        const bool enable_weight_computation,
        const double timescale_bound,
        const int walk_padding_value): use_gpu(use_gpu) {
    temporal_random_walk = new TemporalRandomWalkStore(
        is_directed,
        use_gpu,
        max_time_capacity,
        enable_weight_computation,
        timescale_bound,
        walk_padding_value);
}

TemporalRandomWalk::~TemporalRandomWalk() {
    delete temporal_random_walk;
}

void TemporalRandomWalk::add_multiple_edges(
    const int* sources,
    const int* targets,
    const int64_t* timestamps,
    const size_t edges_size) const {
    temporal_random_walk::add_multiple_edges(
        temporal_random_walk,
        sources,
        targets,
        timestamps,
        edges_size);
}

void TemporalRandomWalk::add_multiple_edges(const std::vector<std::tuple<int, int, int64_t>>& edges) const {
    std::vector<int> sources;
    std::vector<int> targets;
    std::vector<int64_t> timestamps;

    sources.reserve(edges.size());
    targets.reserve(edges.size());
    timestamps.reserve(edges.size());

    for (const auto& edge : edges) {
        sources.push_back(std::get<0>(edge));
        targets.push_back(std::get<1>(edge));
        timestamps.push_back(std::get<2>(edge));
    }

    add_multiple_edges(sources.data(), targets.data(), timestamps.data(), timestamps.size());
}

WalkSet TemporalRandomWalk::get_random_walks_and_times_for_all_nodes(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_per_node,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {
    WalkSet walk_set;

    #ifdef HAS_CUDA
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    }
    else
    #endif
    {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    }

    return walk_set;
}

WalkSet TemporalRandomWalk::get_random_walks_and_times(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_total,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    WalkSet walk_set;

    #ifdef HAS_CUDA
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    }
    else
    #endif
    {
        walk_set = temporal_random_walk::get_random_walks_and_times_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    }

    return walk_set;
}

size_t TemporalRandomWalk::get_node_count() const {
    return temporal_random_walk::get_node_count(temporal_random_walk);
}

size_t TemporalRandomWalk::get_edge_count() const {
    #ifdef HAS_CUDA
    if (use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        CUDA_CHECK_AND_CLEAR(hipMalloc(&d_result, sizeof(size_t)));

        TemporalRandomWalkStore* d_temporal_random_walk = temporal_random_walk::to_device_ptr(temporal_random_walk);
        get_edge_count_kernel<<<1, 1>>>(d_result, d_temporal_random_walk);
        CUDA_KERNEL_CHECK("After get_edge_count_kernel execution");

        size_t host_result;
        CUDA_CHECK_AND_CLEAR(hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost));

        CUDA_CHECK_AND_CLEAR(hipFree(d_result));
        temporal_random_walk::free_device_pointers(d_temporal_random_walk);

        return host_result;
    }
    else
    #endif
    {
        // Direct call for CPU implementation
        return temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

std::vector<int> TemporalRandomWalk::get_node_ids() const {
    const DataBlock<int> node_ids = temporal_random_walk::get_node_ids(temporal_random_walk);
    std::vector<int> result;

    #ifdef HAS_CUDA
    if (node_ids.use_gpu) {
        // Allocate temporary host memory
        int* host_data = new int[node_ids.size];
        CUDA_CHECK_AND_CLEAR(hipMemcpy(host_data, node_ids.data,
                                     node_ids.size * sizeof(int),
                                     hipMemcpyDeviceToHost));

        result.assign(host_data, host_data + node_ids.size);

        delete[] host_data;
    }
    else
    #endif
    {
        result.assign(node_ids.data, node_ids.data + node_ids.size);
    }

    return result;
}

std::vector<std::tuple<int, int, int64_t>> TemporalRandomWalk::get_edges() const {
    const DataBlock<Edge> edges = temporal_random_walk::get_edges(temporal_random_walk);
    std::vector<std::tuple<int, int, int64_t>> result;
    result.reserve(edges.size);

    #ifdef HAS_CUDA
    if (edges.use_gpu) {
        auto host_edges = new Edge[edges.size];
        CUDA_CHECK_AND_CLEAR(hipMemcpy(host_edges, edges.data,
                                    edges.size * sizeof(Edge),
                                    hipMemcpyDeviceToHost));

        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(
                host_edges[i].u,
                host_edges[i].i,
                host_edges[i].ts);
        }

        delete[] host_edges;
    }
    else
    #endif
    {
        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(
                edges.data[i].u,
                edges.data[i].i,
                edges.data[i].ts);
        }
    }

    return result;
}

bool TemporalRandomWalk::get_is_directed() const {
    return temporal_random_walk::get_is_directed(temporal_random_walk);
}

void TemporalRandomWalk::clear() const {
    temporal_random_walk::clear(temporal_random_walk);
}

size_t TemporalRandomWalk::get_memory_used() const {
    return temporal_random_walk::get_memory_used(temporal_random_walk);
}
