#include "hip/hip_runtime.h"
#include "TemporalRandomWalkProxy.cuh"

__global__ void get_edge_count_kernel(size_t* result, const TemporalRandomWalk* temporal_random_walk) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

TemporalRandomWalkProxy::TemporalRandomWalkProxy(
        const bool is_directed,
        const bool use_gpu,
        const int64_t max_time_capacity,
        const bool enable_weight_computation,
        const double timescale_bound,
        const size_t n_threads): use_gpu(use_gpu) {
    temporal_random_walk = new TemporalRandomWalk(is_directed, use_gpu, max_time_capacity, enable_weight_computation, timescale_bound, n_threads);
}

TemporalRandomWalkProxy::~TemporalRandomWalkProxy() {
    delete temporal_random_walk;
}

void TemporalRandomWalkProxy::add_multiple_edges(const std::vector<std::tuple<int, int, int64_t>>& edges) const {
    Edge* edge_array = new Edge[edges.size()];
    for (size_t idx = 0; idx < edges.size(); idx++) {
        const auto& [u, i, ts] = edges[idx];
        edge_array[idx] = Edge(u, i, ts);
    }

    temporal_random_walk::add_multiple_edges(temporal_random_walk, edge_array, edges.size());

    delete[] edge_array;
}

std::vector<std::vector<NodeWithTime>> TemporalRandomWalkProxy::get_random_walks_and_times_for_all_nodes(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_per_node,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    WalkSet walk_set;
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    } else {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_per_node,
            initial_edge_bias,
            walk_direction);
    }

    std::vector<std::vector<NodeWithTime>> walks(walk_set.num_walks);
    for (size_t walk_idx = 0; walk_idx < walk_set.num_walks; walk_idx++) {
        const size_t walk_len = walk_set.get_walk_len(static_cast<int>(walk_idx));

        walks[walk_idx].reserve(walk_len);

        for (size_t hop = 0; hop < walk_len; hop++) {
            NodeWithTime node_time = walk_set.get_walk_hop(static_cast<int>(walk_idx), static_cast<int>(hop));
            walks[walk_idx].push_back(node_time);
        }
    }

    std::vector<std::vector<NodeWithTime>> non_empty_walks;
    std::copy_if(walks.begin(), walks.end(), std::back_inserter(non_empty_walks),
                 [](const std::vector<NodeWithTime>& v) { return !v.empty(); });

    return non_empty_walks;
}

std::vector<std::vector<int>> TemporalRandomWalkProxy::get_random_walks_for_all_nodes(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_per_node,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    auto walks_with_times = get_random_walks_and_times_for_all_nodes(
        max_walk_len, walk_bias, num_walks_per_node, initial_edge_bias, walk_direction);

    std::vector<std::vector<int>> result(walks_with_times.size());
    for (size_t i = 0; i < walks_with_times.size(); i++) {
        result[i].reserve(walks_with_times[i].size());
        for (const auto& node_time : walks_with_times[i]) {
            result[i].push_back(node_time.node);
        }
    }

    return result;
}

std::vector<std::vector<NodeWithTime>> TemporalRandomWalkProxy::get_random_walks_and_times(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_total,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    WalkSet walk_set;
    if (use_gpu) {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_cuda(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    } else {
        walk_set = temporal_random_walk::get_random_walks_and_times_for_all_nodes_std(
            temporal_random_walk,
            max_walk_len,
            walk_bias,
            num_walks_total,
            initial_edge_bias,
            walk_direction);
    }

    std::vector<std::vector<NodeWithTime>> walks(walk_set.num_walks);
    for (size_t walk_idx = 0; walk_idx < walk_set.num_walks; walk_idx++) {
        const size_t walk_len = walk_set.get_walk_len(static_cast<int>(walk_idx));

        walks[walk_idx].reserve(walk_len);

        for (size_t hop = 0; hop < walk_len; hop++) {
            NodeWithTime node_time = walk_set.get_walk_hop(static_cast<int>(walk_idx), static_cast<int>(hop));
            walks[walk_idx].push_back(node_time);
        }
    }

    std::vector<std::vector<NodeWithTime>> non_empty_walks;
    std::copy_if(walks.begin(), walks.end(), std::back_inserter(non_empty_walks),
                 [](const std::vector<NodeWithTime>& v) { return !v.empty(); });

    return non_empty_walks;
}

std::vector<std::vector<int>> TemporalRandomWalkProxy::get_random_walks(
        const int max_walk_len,
        const RandomPickerType* walk_bias,
        const int num_walks_total,
        const RandomPickerType* initial_edge_bias,
        const WalkDirection walk_direction) const {

    auto walks_with_times = get_random_walks_and_times_for_all_nodes(
        max_walk_len, walk_bias, num_walks_total, initial_edge_bias, walk_direction);

    std::vector<std::vector<int>> result(walks_with_times.size());
    for (size_t i = 0; i < walks_with_times.size(); i++) {
        result[i].reserve(walks_with_times[i].size());
        for (const auto& node_time : walks_with_times[i]) {
            result[i].push_back(node_time.node);
        }
    }

    return result;
}

size_t TemporalRandomWalkProxy::get_node_count() const {
    return temporal_random_walk::get_node_count(temporal_random_walk);
}

size_t TemporalRandomWalkProxy::get_edge_count() const {
    if (use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        hipMalloc(&d_result, sizeof(size_t));

        TemporalRandomWalk* d_temporal_random_walk = temporal_random_walk::to_device_ptr(temporal_random_walk);
        get_edge_count_kernel<<<1, 1>>>(d_result, temporal_random_walk);

        size_t host_result;
        hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_temporal_random_walk);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return temporal_random_walk::get_edge_count(temporal_random_walk);
    }
}

std::vector<int> TemporalRandomWalkProxy::get_node_ids() const {
    const DataBlock<int> node_ids = temporal_random_walk::get_node_ids(temporal_random_walk);
    std::vector<int> result;

    if (node_ids.use_gpu) {
        // Allocate temporary host memory
        int* host_data = new int[node_ids.size];
        hipMemcpy(host_data, node_ids.data,
                                     node_ids.size * sizeof(int),
                                     hipMemcpyDeviceToHost);

        result.assign(host_data, host_data + node_ids.size);

        delete[] host_data;
    } else {
        result.assign(node_ids.data, node_ids.data + node_ids.size);
    }

    return result;
}

std::vector<std::tuple<int, int, int64_t>> TemporalRandomWalkProxy::get_edges() const {
    const DataBlock<Edge> edges = temporal_random_walk::get_edges(temporal_random_walk);
    std::vector<std::tuple<int, int, int64_t>> result;
    result.reserve(edges.size);

    if (edges.use_gpu) {
        auto host_edges = new Edge[edges.size];
        hipMemcpy(host_edges, edges.data,
                                    edges.size * sizeof(Edge),
                                    hipMemcpyDeviceToHost);

        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(host_edges[i].u, host_edges[i].i, host_edges[i].ts);
        }

        delete[] host_edges;
    } else {
        for (size_t i = 0; i < edges.size; i++) {
            result.emplace_back(edges.data[i].u, edges.data[i].i, edges.data[i].ts);
        }
    }

    return result;
}

bool TemporalRandomWalkProxy::get_is_directed() const {
    return temporal_random_walk::get_is_directed(temporal_random_walk);
}

void TemporalRandomWalkProxy::clear() const {
    temporal_random_walk::clear(temporal_random_walk);
}
