#include "hip/hip_runtime.h"
#include "EdgeDataProxy.cuh"

__global__ void empty_kernel(bool* result, const EdgeData* edge_data) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::empty(edge_data);
    }
}

__global__ void size_kernel(size_t* result, const EdgeData* edge_data) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::size(edge_data);
    }
}


__global__ void find_group_after_timestamp_kernel(size_t* result, const EdgeData* edge_data, int64_t timestamp) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::find_group_after_timestamp_device(edge_data, timestamp);
    }
}

__global__ void find_group_before_timestamp_kernel(size_t* result, const EdgeData* edge_data, int64_t timestamp) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::find_group_before_timestamp_device(edge_data, timestamp);
    }
}

__global__ void get_timestamp_group_range_kernel(SizeRange* result, const EdgeData* edge_data, size_t group_idx) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::get_timestamp_group_range(edge_data, group_idx);
    }
}

__global__ void get_timestamp_group_count_kernel(size_t* result, const EdgeData* edge_data) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = edge_data::get_timestamp_group_count(edge_data);
    }
}

EdgeDataProxy::EdgeDataProxy(const bool use_gpu): owns_edge_data(true) {
    edge_data = new EdgeData(use_gpu);
}

EdgeDataProxy::EdgeDataProxy(EdgeData* existing_edge_data) : edge_data(existing_edge_data), owns_edge_data(false) {}

EdgeDataProxy::~EdgeDataProxy() {
    if (owns_edge_data && edge_data) {
        delete edge_data;
    }
}

void EdgeDataProxy::reserve(size_t size) const {
    edge_data::reserve(edge_data, size);
}

void EdgeDataProxy::clear() const {
    edge_data::clear(edge_data);
}

size_t EdgeDataProxy::size() const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        hipMalloc(&d_result, sizeof(size_t));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        size_kernel<<<1, 1>>>(d_result, d_edge_data);

        size_t host_result;
        hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return edge_data::size(edge_data);
    }
}

bool EdgeDataProxy::empty() const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        bool* d_result;
        hipMalloc(&d_result, sizeof(bool));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        empty_kernel<<<1, 1>>>(d_result, d_edge_data);

        bool host_result;
        hipMemcpy(&host_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return edge_data::empty(edge_data);
    }
}

void EdgeDataProxy::add_edges(const std::vector<int>& sources, const std::vector<int>& targets, const std::vector<int64_t>& timestamps) const {
    if (sources.size() != targets.size() || sources.size() != timestamps.size()) {
        throw std::runtime_error("Vector sizes don't match for add_edges");
    }

    const size_t size = sources.size();

    if (edge_data->use_gpu) {
        // Allocate GPU memory for the data
        int* d_sources = nullptr;
        int* d_targets = nullptr;
        int64_t* d_timestamps = nullptr;

        hipMalloc(&d_sources, size * sizeof(int));
        hipMalloc(&d_targets, size * sizeof(int));
        hipMalloc(&d_timestamps, size * sizeof(int64_t));

        // Copy data to GPU
        hipMemcpy(d_sources, sources.data(), size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_targets, targets.data(), size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_timestamps, timestamps.data(), size * sizeof(int64_t), hipMemcpyHostToDevice);

        // Call add_edges
        edge_data::add_edges(edge_data, d_sources, d_targets, d_timestamps, size);

        // Free GPU memory
        hipFree(d_sources);
        hipFree(d_targets);
        hipFree(d_timestamps);
    } else {
        // Direct call for CPU implementation
        edge_data::add_edges(edge_data, sources.data(), targets.data(), timestamps.data(), size);
    }
}

void EdgeDataProxy::push_back(const int source, const int target, const int64_t timestamp) const {
    if (edge_data->use_gpu) {
        // Allocate GPU memory for single elements
        int* d_source = nullptr;
        int* d_target = nullptr;
        int64_t* d_timestamp = nullptr;

        hipMalloc(&d_source, sizeof(int));
        hipMalloc(&d_target, sizeof(int));
        hipMalloc(&d_timestamp, sizeof(int64_t));

        // Copy data to GPU
        hipMemcpy(d_source, &source, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_target, &target, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_timestamp, &timestamp, sizeof(int64_t), hipMemcpyHostToDevice);

        // Call add_edges with single element
        edge_data::add_edges(edge_data, d_source, d_target, d_timestamp, 1);

        // Free GPU memory
        hipFree(d_source);
        hipFree(d_target);
        hipFree(d_timestamp);
    } else {
        // For CPU implementation, create small arrays
        const int sources[1] = { source };
        const int targets[1] = { target };
        const int64_t timestamps[1] = { timestamp };

        edge_data::add_edges(edge_data, sources, targets, timestamps, 1);
    }
}

std::vector<Edge> EdgeDataProxy::get_edges() const {
    // Call the optimized edge_data::get_edges function directly
    const DataBlock<Edge> edges_block = edge_data::get_edges(edge_data);
    std::vector<Edge> result;

    // Copy data from DataBlock to std::vector
    if (edge_data->use_gpu) {
        // For GPU data, need to copy from device to host
        const auto host_edges = new Edge[edges_block.size];
        hipMemcpy(host_edges, edges_block.data, edges_block.size * sizeof(Edge), hipMemcpyDeviceToHost);

        result.assign(host_edges, host_edges + edges_block.size);
        delete[] host_edges;

        // Free device memory for DataBlock
        if (edges_block.data) {
            hipFree(edges_block.data);
        }
    } else {
        // For CPU data, can directly copy
        result.assign(edges_block.data, edges_block.data + edges_block.size);

        // Free host memory for DataBlock
        delete[] edges_block.data;
    }

    return result;
}

void EdgeDataProxy::update_timestamp_groups() const {
    if (edge_data->use_gpu) {
        edge_data::update_timestamp_groups_cuda(edge_data);
    } else {
        edge_data::update_timestamp_groups_std(edge_data);
    }
}

void EdgeDataProxy::update_temporal_weights(double timescale_bound) const {
    if (edge_data->use_gpu) {
        edge_data::update_temporal_weights_cuda(edge_data, timescale_bound);
    } else {
        edge_data::update_temporal_weights_std(edge_data, timescale_bound);
    }
}

std::pair<size_t, size_t> EdgeDataProxy::get_timestamp_group_range(size_t group_idx) const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        SizeRange* d_result;
        hipMalloc(&d_result, sizeof(SizeRange));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        get_timestamp_group_range_kernel<<<1, 1>>>(d_result, d_edge_data, group_idx);

        SizeRange host_result;
        hipMemcpy(&host_result, d_result, sizeof(SizeRange), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return {host_result.from, host_result.to};
    } else {
        // Direct call for CPU implementation
        SizeRange result = edge_data::get_timestamp_group_range(edge_data, group_idx);
        return {result.from, result.to};
    }
}

size_t EdgeDataProxy::get_timestamp_group_count() const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        hipMalloc(&d_result, sizeof(size_t));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        get_timestamp_group_count_kernel<<<1, 1>>>(d_result, d_edge_data);

        size_t host_result;
        hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return edge_data::get_timestamp_group_count(edge_data);
    }
}

size_t EdgeDataProxy::find_group_after_timestamp(int64_t timestamp) const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        hipMalloc(&d_result, sizeof(size_t));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        find_group_after_timestamp_kernel<<<1, 1>>>(d_result, d_edge_data, timestamp);

        size_t host_result;
        hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return edge_data::find_group_after_timestamp(edge_data, timestamp);
    }
}

size_t EdgeDataProxy::find_group_before_timestamp(int64_t timestamp) const {
    if (edge_data->use_gpu) {
        // Call via CUDA kernel for GPU implementation
        size_t* d_result;
        hipMalloc(&d_result, sizeof(size_t));

        EdgeData* d_edge_data = edge_data::to_device_ptr(edge_data);
        find_group_before_timestamp_kernel<<<1, 1>>>(d_result, d_edge_data, timestamp);

        size_t host_result;
        hipMemcpy(&host_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);

        hipFree(d_result);
        hipFree(d_edge_data);

        return host_result;
    } else {
        // Direct call for CPU implementation
        return edge_data::find_group_before_timestamp(edge_data, timestamp);
    }
}
